#include "hip/hip_runtime.h"
#include "CalCulation.cuh"

//��λ������
__device__ Point C_UnitizeVectorOnGPU(Point vector)
{
	float sum=sqrt(pow(vector.x,2)+pow(vector.y,2)+pow(vector.z,2));
	Point unitVector;
	unitVector.x=vector.x/sum;
	unitVector.y=vector.y/sum;
	unitVector.z=vector.z/sum;
	return unitVector;
}

//��������
__device__ Point GetReverseVector(Point vector)
{
	vector.x=-vector.x;
	vector.y=-vector.y;
	vector.z=-vector.z;
	return vector;
}

//������֮�����
__device__ float CalculateDistanceOfBipartiteOnGPU(Point p1, Point p2)
{
	return sqrt(pow(p1.x-p2.x, 2) + pow(p1.y-p2.y, 2) + pow(p1.z-p2.z, 2));
}


//��һ���������һ����ĶԳƵ�
__device__ Point GetSymmetryPoint(Point pointToCal, Point centerPoint)
{
	Point result;
	result.x= 2*centerPoint.x-pointToCal.x;
	result.y= 2*centerPoint.y-pointToCal.y;
	result.z= 2*centerPoint.z-pointToCal.z;
	return result;
}

//��������ģ
__device__ float Mag(Point vector)
{
	return sqrt(pow(vector.x,2)+pow(vector.y,2)+pow(vector.z,2));
}

//����
__device__ float DotMultiply(Point vector1, Point vector2)
{
	return vector1.x*vector2.x+vector1.y*vector2.y+vector1.z*vector2.z;
}

//����
__device__ Point CrossMultiply(Point vector1, Point vector2)
{
	Point result;
	float xTemp, yTemp, zTemp, multipled;
	xTemp = vector1.y*vector2.z-vector1.z*vector2.y;
	yTemp = vector1.z*vector2.x-vector1.x*vector2.z;
	zTemp = vector1.x*vector2.y-vector2.y*vector1.x;
	multipled = sqrt(pow(xTemp,2)+pow(yTemp,2)+pow(zTemp,2));
	if(multipled==0)
	{
		result.x=0;
		result.y=0;
		result.z=0;
	}
	else
	{
		result.x=xTemp/multipled;
		result.y=yTemp/multipled;
		result.z=zTemp/multipled;
	}
	if(abs(result.x)<0.000001)
		result.x=0;
	if(abs(result.y)<0.000001)
		result.y=0;
	if(abs(result.z)<0.000001)
		result.z=0;
	return result;
}

//���ƽ�淨����
__device__ Point GetNormalVector(Face face)
{
	Point vectorAB;
	Point vectorBC;
	vectorAB.x=face.B.x-face.A.x;
	vectorAB.y=face.B.y-face.A.y;
	vectorAB.z=face.B.z-face.A.z;
	vectorBC.x=face.C.x-face.B.x;
	vectorBC.y=face.C.y-face.B.y;
	vectorBC.z=face.C.z-face.B.z;
	return CrossMultiply(vectorAB,vectorBC);
}

//��ƽ�淽��Ax+By+Cz+D=0���ĸ�ϵ��A,B,C,D
__device__ float* GetFaceEquationFactor(Face face)
{
	float para[4];
	Point unitVector=GetNormalVector(face);
	unitVector=C_UnitizeVectorOnGPU(unitVector);
	para[0]=unitVector.x;
	para[1]=unitVector.y;
	para[2]=unitVector.z;
	para[3]=-(unitVector.x*face.A.x+unitVector.y*face.A.y+unitVector.z*face.A.z);
	return para;
}

//�жϵ��Ƿ���ƽ����
__device__ bool JudgeIfPointInFace(Point point, Face face)
{
	float* faceFactor=(float*)malloc(4);//ƽ�淽�̵�4��ϵ��ABCD
	faceFactor=GetFaceEquationFactor(face);
	float faceEquation=faceFactor[0]*point.x+faceFactor[1]*point.y+faceFactor[2]*point.z+faceFactor[3];
	if(abs(faceEquation<0.001))
	{
		delete [] faceFactor;
		return true;
	}
	else
	{
		delete [] faceFactor;
		return false;
	}
}

//�ж����������Ƿ�ƽ����ͬ��
__device__ bool IsParallelAndSamedirection(Point vector1, Point vector2)
{
	float temp=(vector1.x*vector2.x+vector1.y*vector2.y+vector1.z*vector2.z)/(Mag(vector1)*Mag(vector2));
	if(abs(temp-1)<=0.00001)
	{
		return true;
	}
	else
	{
		return false;
	}
}

//�õ�һ����ķ���������p1Ϊ�µ㣬p2Ϊ�ϵ㣬p1ָ��p2��
__device__ Point GetDirectionVectorOfEdge(Edge edge)
{
	Point direction;
	direction.x=edge.p2.x-edge.p1.x;
	direction.y=edge.p2.y-edge.p1.y;
	direction.z=edge.p2.z-edge.p1.z;
	return direction;
}

//�ж��������Ƿ�Ϊͬһ��
__device__ bool IsTwoPointSame(Point A,Point B)
{
	if ((abs(A.x-B.x)<0.00001)&&(abs(A.y-B.y)<0.00001)&&(abs(A.z-B.z)<0.00001))
	{
		return true;
	}
	return false;
}

//�ж�һ�����Ƿ�Ϊһ��������Ķ���֮һ
__device__ bool IsTheSamePointOfAdjacentExist(Point A,Face face)
{
	if (IsTwoPointSame(A,face.A)||IsTwoPointSame(A,face.B)||IsTwoPointSame(A,face.C))
	{
		return true;
	}
	return false;
}

//�õ�������������Ĺ�����
__device__ Edge GetEdgeOfAdjacentFaces(Face face1,Face face2)
{
	Edge edge;
	if(IsTheSamePointOfAdjacentExist(face1.A,face2)&&IsTheSamePointOfAdjacentExist(face1.B,face2)&&IsTheSamePointOfAdjacentExist(face1.C,face2))
	{
		//��ʾ��������Ϊͬһ������
		edge.p1.x=6666;
		edge.p1.y=6666;
		edge.p1.z=6666;
		edge.p2.x=6666;
		edge.p2.y=6666;
		edge.p2.z=6666;
		return edge;
	}
	else if (IsTheSamePointOfAdjacentExist(face1.A,face2)&&IsTheSamePointOfAdjacentExist(face1.B,face2))
	{
		edge.p1=face1.A;
		edge.p2=face1.B;
		return edge;
	}
	else if (IsTheSamePointOfAdjacentExist(face1.A,face2)&&IsTheSamePointOfAdjacentExist(face1.C,face2))
	{
		edge.p1=face1.A;
		edge.p2=face1.C;
		return edge;
	}
	else if (IsTheSamePointOfAdjacentExist(face1.B,face2)&&IsTheSamePointOfAdjacentExist(face1.C,face2))
	{
		edge.p1=face1.B;
		edge.p2=face1.C;
		return edge;
	}
	else
	{
		//��ʾ���������޹�����
		edge.p1.x=8888;
		edge.p1.y=8888;
		edge.p1.z=8888;
		edge.p2.x=8888;
		edge.p2.y=8888;
		edge.p2.z=8888;
		return edge;
	}
}

//���ֱ����һ�㵽��ֱ�ߵĴ��㣬pΪֱ����һ�㣬���ش�������
__device__ Point GetDropFoot(Point p)
{
	
}


//����������г���һ�������ʣ��һ��
__device__ Point GetRemainPointOnFace(Face face, Edge sameEdge)
{
	bool flag[3];
	for(int i=0;i<3;i++)
	{
		flag[i]=true;
	}
	//����6��if��������Ϊ�棬����ʣ��һ��
	if(IsTwoPointSame(face.A, sameEdge.p1))
	{
		flag[0]=false;
	}
	if(IsTwoPointSame(face.B, sameEdge.p1))
	{
		flag[1]=false;
	}
	if(IsTwoPointSame(face.C, sameEdge.p1))
	{
		flag[2]=false;
	}
	if(IsTwoPointSame(face.A, sameEdge.p2))
	{
		flag[0]=false;
	}
	if(IsTwoPointSame(face.B, sameEdge.p2))
	{
		flag[1]=false;
	}
	if(IsTwoPointSame(face.C, sameEdge.p2))
	{
		flag[2]=false;
	}
	if(flag[0])
	{
		return face.A;
	}
	if(flag[1])
	{
		return face.B;
	}
	if(flag[2])
	{
		return face.C;
	}
	printf("���������г��������һ��GetRemainPointOnFace()����\n");
}

//�����ⷢ��㵽��������¹�����������
__device__ Ray CreateNewInRay(Point originPoint, Point diffractionPoint)
{
	Ray newInRay;
	newInRay.originalPoint=originPoint;
	newInRay.direction.x=diffractionPoint.x-originPoint.x;
	newInRay.direction.y=diffractionPoint.y-originPoint.y;
	newInRay.direction.z=diffractionPoint.z-originPoint.z;
	return newInRay;
}

//������ֱ�ߵļн� �����������ڻ�ȡ���������ߵļн�
__device__ float GetAngleOfTwoStraightLines(Point vector1, Point vector2)
{
	float dotMultiplied= DotMultiply(vector1,vector2);
	float tempValue= sqrt(pow(vector1.x,2)+pow(vector1.y,2)+pow(vector1.z,2))*sqrt(pow(vector2.x,2)+pow(vector2.y,2)+pow(vector2.z,2));
	if(tempValue==0)
	{
		//��Ϊ���ܴ�LOG���˴����Լ�����
		return 90;
	}
	float nTemp=dotMultiplied/tempValue;
	if(nTemp<-1)
	{
		nTemp=-1;
	}
	if(nTemp>1)
	{
		nTemp=1;
	}
	float phase = (acos(nTemp)/PI)*(180.0);
	return phase;
}

//������Բ��Բ�� ��������������������ѡһ������������ͬ���Ҳ��������ĵ�,һ��ѡ�������ĳ������,���н�Ϊ90��ʱ����������ΪԲ�̣�Բ��Ϊ�����
__device__ Point GetCircleCenterPoint(float angleOfRayAndSameLine,Ray incidentRay, Edge sameEdge, Point diffractionPoint)
{
	Point sameEdgePoint1=sameEdge.p1;
	Point sameEdgePoint2=sameEdge.p2;
	if(abs(angleOfRayAndSameLine-90)<0.0001)//�����������ֱ
	{
		return diffractionPoint;
	}
	//��������պ�Ϊ������Ķ��㣬�򱶳������������ߣ���������������ֱ�������ߣ�����ΪԲ��
	else if(IsTwoPointSame(diffractionPoint,sameEdge.p1)||IsTwoPointSame(diffractionPoint,sameEdge.p2))
	{
		if(IsTwoPointSame(diffractionPoint,sameEdge.p1))
		{
			Point p;
			p=GetSymmetryPoint(incidentRay.originalPoint, sameEdge.p1);//������
			//�������򹫹�������ֱ��������
			Point origin;
			origin.x=sameEdge.p2.x;
			origin.y=sameEdge.p2.y;
			origin.z=sameEdge.p2.z;
			Point direction;
			direction.x=sameEdge.p1.x-sameEdge.p2.x;
			direction.y=sameEdge.p1.y-sameEdge.p2.y;
			direction.z=sameEdge.p1.z-sameEdge.p2.z;
			Point end;
			end.x=origin.x+direction.x*100;
			end.y=origin.y+direction.y*100;
			end.z=origin.z+direction.z*100;
			float dx = origin.x-end.x;
			float dy = origin.y-end.y;
			float dz = origin.z-end.z;
			float u = (p.x - origin.x) * (origin.x - end.x) + (p.y - origin.y) * (origin.y - end.y) + (p.z - origin.z) * (origin.z - end.z);
            u = u / ((dx * dx) + (dy * dy) + (dz * dz));
			Point result;
			result.x = origin.x + u*dx;
			result.y = origin.y + u*dy;
			result.z = origin.z + u*dz;
			return result;
		}
		if(IsTwoPointSame(diffractionPoint,sameEdge.p2))
		{
			Point p;
			p=GetSymmetryPoint(incidentRay.originalPoint, sameEdge.p2);//������
			//�������򹫹�������ֱ��������
			Point origin;
			origin.x=sameEdge.p1.x;
			origin.y=sameEdge.p1.y;
			origin.z=sameEdge.p1.z;
			Point direction;
			direction.x=sameEdge.p2.x-sameEdge.p1.x;
			direction.y=sameEdge.p2.y-sameEdge.p1.y;
			direction.z=sameEdge.p2.z-sameEdge.p1.z;
			Point end;
			end.x=origin.x+direction.x*100;
			end.y=origin.y+direction.y*100;
			end.z=origin.z+direction.z*100;
			float dx = origin.x-end.x;
			float dy = origin.y-end.y;
			float dz = origin.z-end.z;
			float u = (p.x - origin.x) * (origin.x - end.x) + (p.y - origin.y) * (origin.y - end.y) + (p.z - origin.z) * (origin.z - end.z);
            u = u / ((dx * dx) + (dy * dy) + (dz * dz));
			Point result;
			result.x = origin.x + u*dx;
			result.y = origin.y + u*dy;
			result.z = origin.z + u*dz;
			return result;
		}
	}
	else//��Բ����Ϊ����������ͬһ��Ķ���
	{
		Point sameEdgeVector;
		sameEdgeVector.x=sameEdgePoint2.x-sameEdgePoint1.x;
		sameEdgeVector.y=sameEdgePoint2.y-sameEdgePoint1.y;
		sameEdgeVector.z=sameEdgePoint2.z-sameEdgePoint1.z;
		float dotMultiplied=DotMultiply(incidentRay.direction, sameEdgeVector);
		if(dotMultiplied>0)//������ͬ��
		{
			return sameEdgePoint2;
		}
		if(dotMultiplied<0)//����������
		{
			return sameEdgePoint1;
		}
	}
}

//������Բ�İ뾶
__device__ float GetCircleRadius(float angleOfRayAndSameLine, Point diffractionPoint, Point circleCenterPoint)
{
	float circleRadius;
	if(abs(angleOfRayAndSameLine-90)<0.0001)//���ߴ�ֱ������
	{
		circleRadius=1;
	}
	else
	{
		//tan��Ϊ�۽�ʱ���п��ܳ��ָ������������������������ߵķ������������������ԼӾ���ֵ
		circleRadius=abs(CalculateDistanceOfBipartiteOnGPU(diffractionPoint,circleCenterPoint)*tan(angleOfRayAndSameLine*PI/180));
	}
	return circleRadius;
}

//��Բ����ƽ���ϵ�һ������(����) �������Ϊ��������������е�����һ����
__device__ Point GetVectorInThePlaneOfCircle(Edge sameEdge, Face oneAdjacentTriangle)
{
	/*Point diffractionEdgePoint1 = sameEdge.p1;
	Point diffractionEdgePoint2 = sameEdge.p2;
	if(diffractionEdgePoint2.x-diffractionEdgePoint1.x==0 && diffractionEdgePoint2.y-diffractionEdgePoint1.y==0)
	{
		Point unitCircleVectorU;
		unitCircleVectorU.x=1;
		unitCircleVectorU.y=0;
		unitCircleVectorU.z=0;
		return unitCircleVectorU;
	}
	else
	{*/
		return C_UnitizeVectorOnGPU(GetNormalVector(oneAdjacentTriangle));
	//}
}

//����Բ��ƽ���ϵ�һ����ƽ�淨������ƽ��ĳ������������������
__device__ Point GetAnotherVectorInThePlaneOfCircle(Edge sameEdge, Point vectorU)
{
	Point diffractionEdgePoint1 = sameEdge.p1;
	Point diffractionEdgePoint2 = sameEdge.p2;
	Point sameEdgeVector;
	sameEdgeVector.x=diffractionEdgePoint2.x-diffractionEdgePoint1.x;
	sameEdgeVector.y=diffractionEdgePoint2.y-diffractionEdgePoint1.y;
	sameEdgeVector.z=diffractionEdgePoint2.z-diffractionEdgePoint1.z;
	/*if(diffractionEdgePoint2.x-diffractionEdgePoint1.x==0 && diffractionEdgePoint2.y-diffractionEdgePoint1.y==0)
	{
		Point unitCircleVectorV;
		unitCircleVectorV.x=0;
		unitCircleVectorV.y=1;
		unitCircleVectorV.z=0;
		return unitCircleVectorV;
	}
	else*/
	{
		//����һ����1������������������������������Ҳ��Բ���ڵ�ƽ����
		Point circleVectorV;
		circleVectorV.x=sameEdgeVector.y*vectorU.z-sameEdgeVector.z*vectorU.y;
		circleVectorV.y=sameEdgeVector.z*vectorU.x-sameEdgeVector.x*vectorU.z;
		circleVectorV.z=sameEdgeVector.x*vectorU.y-sameEdgeVector.y*vectorU.x;
		return C_UnitizeVectorOnGPU(circleVectorV);
	}
}

//��Բ��������Ľ��㣨���䣩
/// <param name="terTri">������</param>
/// <param name="centerPoint">Բ��</param>
/// <param name="radius">Բ�İ뾶</param>
/// <param name="vectorU">Բ����ƽ���һ������U������ƽ��ķ�������ֱ</param>
/// <param name="vectorV">Բ����ƽ���һ������V,����U��ƽ�淨�����໥��ֱ</param>
/// <param name="triPoint">��������������������˵������һ����</param>
/// <returns>����Բ��������Ľ���</returns>
__device__ Point GetCrossPointOfCircleWithTer(Face terTri, Point centerPoint, float radius, Point vectorU, Point vectorV, Point triPoint)
{
	//������ �������õ�һ���ڸ߿յ�����
	Point defaultPoint;
	defaultPoint.x=7777;
	defaultPoint.y=7777;
	defaultPoint.z=7777;
	//�ο�Rays.cs�е�ͬ�����������巽���ڸöδ�����
	Point crossPoints[4];
	bool crossPointFlags[4];
	for(int i = 0; i < 4; i++)//����־λ����ֵ��Ĭ��ȫΪtrue��֮��������޳�
	{
		crossPointFlags[i]=true;
	}
	if(radius<1)//�뾶̫С�����ж������������ð뾶��С��1
	{
		radius=1;
	}
	Point triVector= GetNormalVector(terTri);//��ȡ������ķ�����
	float D = -(triVector.x*terTri.A.x+triVector.y*terTri.A.y+triVector.z*terTri.A.z);
	float M = radius*(triVector.x*vectorU.x+triVector.y*vectorU.y+triVector.z*vectorU.z);
	float N = radius*(triVector.x*vectorV.x+triVector.y*vectorV.y+triVector.z*vectorV.z);
	float sint[2];
	sint[0]=round(sqrt(pow(N,2)/(pow(M,2)+pow(N,2))));
	sint[1]=round(-sqrt(pow(N,2)/(pow(M,2)+pow(N,2))));
	for(int i=0; i<2; i++)//���ڷֱ治��Ƕ�t�ķ�Χ�����Խ�cost��1-sint^2�����ķ����󣬲��õ�4���㣬������������ƽ����
	{
		if((-1<=sint[i])&&(sint[i]<=1))//��ķ�Χ��-1��1֮��ʱ
		{
			
			float cost[2];
			cost[0]=sqrt(1-pow(sint[i],2));
			cost[1]=-sqrt(1-pow(sint[i],2));
			Point crossPoint1;
			crossPoint1.x=centerPoint.x+radius*(vectorU.x*sint[i]+vectorV.x*cost[0]);
			crossPoint1.y=centerPoint.y+radius*(vectorU.y*sint[i]+vectorV.y*cost[0]);
			crossPoint1.z=centerPoint.z+radius*(vectorU.z*sint[i]+vectorV.z*cost[0]);
			crossPoints[2*i] = crossPoint1;
			Point crossPoint2;
			crossPoint2.x=centerPoint.x+radius*(vectorU.x*sint[i]+vectorV.x*cost[1]);
			crossPoint2.y=centerPoint.y+radius*(vectorU.y*sint[i]+vectorV.y*cost[1]);
			crossPoint2.z=centerPoint.z+radius*(vectorU.z*sint[i]+vectorV.z*cost[1]);
			crossPoints[2*i + 1] = crossPoint2;
		}
		else
		{
			//����
			printf("��Բ�������潻��ʱ����\n");
			return defaultPoint;
		}
	}
	//ɾ������������ƽ���ϵĽ���
	for(int i=0;i<4;i++)
	{
		if(!JudgeIfPointInFace(crossPoints[i], terTri))
		{
			crossPointFlags[i] = false;
		}
	}
	//���ڵ�thi��Ϊ90��ʱ����õĵ������������ظ��ĵ㣬��Ҫ����ɾ��
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3-i;j++)
		{
			if(IsTwoPointSame(crossPoints[j], crossPoints[j+1]))
			{
				if(crossPointFlags[j]&&crossPointFlags[j+1])//��ͬ�ĵ��޳�����һ��
				{
					crossPointFlags[j+1]=false;
				}
			}
		}
	}
	//Բ���������������һ��Ľ���
	for(int i=0;i<3;i++)
	{
		for(int j=i+1;j<4;j++)
		{
			if(crossPointFlags[i]&&crossPointFlags[j])//ȡͬ���
			{
				if(CalculateDistanceOfBipartiteOnGPU(crossPoints[i],triPoint)<CalculateDistanceOfBipartiteOnGPU(crossPoints[j],triPoint))
				{
					return crossPoints[i];
				}
				else
				{
					return crossPoints[j];
				}
			}
		}
	}
	//else if((!cp1)&&cp2)
	//{
	//	crossPoint2=GetSymmetryPointOnCPU(crossPoint2, centerPoint);//�Ե�ȡ��
	//	return crossPoint2;
	//}
	{
		//����
		return defaultPoint;
	}
}

//������Բ����������ķֲ���������ϵ
/// <param name="unitVectorU">Բ����ƽ����һ������U</param>
/// <param name="unitVectorV">Բ����ƽ����һ������V</param>
/// <param name="circleCenterPoint">Բ�̵�Բ��</param>
/// <param name="crossPointOfFace0AndCircle">Բ����������0�Ľ���</param>
/// <param name="crossPointOfFace1AndCircle">Բ����������1�Ľ���</param>
/// <param name="verticalVector">Բ����ƽ��ķ����������µ�Բ��ָ���ϵ�Բ��</param>
/// <param name="angleOfTriangles">����������ļн�</param>
__device__ void SetUnitVectorVnCirclePlane(Point &unitVectorU, Point &unitVectorV, Point circleCenterPoint,
	Point crossPointOfFace0AndCircle, Point crossPointOfFace1AndCircle, Point verticalVector, float angleOfTriangles)
{
	unitVectorU.x=crossPointOfFace0AndCircle.x-circleCenterPoint.x;
	unitVectorU.y=crossPointOfFace0AndCircle.y-circleCenterPoint.y;
	unitVectorU.z=crossPointOfFace0AndCircle.z-circleCenterPoint.z;
	unitVectorU=C_UnitizeVectorOnGPU(unitVectorU);
	unitVectorV=C_UnitizeVectorOnGPU(GetReverseVector(CrossMultiply(unitVectorU,verticalVector)));
	if(!IsParallelAndSamedirection(verticalVector,CrossMultiply(unitVectorU,unitVectorV)))
	{
		unitVectorV=GetReverseVector(unitVectorV);
	}
	Point otherVectorU;
	otherVectorU.x=crossPointOfFace1AndCircle.x-circleCenterPoint.x;
	otherVectorU.y=crossPointOfFace1AndCircle.y-circleCenterPoint.y;
	otherVectorU.z=crossPointOfFace1AndCircle.z-circleCenterPoint.z;
	otherVectorU=C_UnitizeVectorOnGPU(otherVectorU);
	if(angleOfTriangles < 90)
	{
		if(abs(GetAngleOfTwoStraightLines(unitVectorV,otherVectorU) + angleOfTriangles - 90)<0.00001)
		{
			unitVectorU = otherVectorU;
			unitVectorV = CrossMultiply(verticalVector,unitVectorU);
		}
	}
	else if(angleOfTriangles>90)
	{
		if(abs(angleOfTriangles - GetAngleOfTwoStraightLines(unitVectorV,otherVectorU) - 90)<0.00001)
		{
			unitVectorU = otherVectorU;
			unitVectorV = CrossMultiply(verticalVector,unitVectorU);
		}
	}
	else
	{
		if(IsParallelAndSamedirection(unitVectorV,otherVectorU))
		{
			unitVectorV = GetReverseVector(unitVectorU);
			unitVectorU=otherVectorU;
		}
	}
}

//��������ĽǶ���Բ�ϵ�һ�� arcAngle�Ի����Ʊ�ʾ
__device__ Point GetPointInCircumference(Point circleCenterPoint, float circleRadius, Point unitVectorU, Point unitVectorV, float arcAngle)
{
	Point circumferencePoint;
	circumferencePoint.x = circleCenterPoint.x + circleRadius * (unitVectorU.x * cos(arcAngle) + unitVectorV.x * sin(arcAngle));
	circumferencePoint.y = circleCenterPoint.y + circleRadius * (unitVectorU.y * cos(arcAngle) + unitVectorV.y * sin(arcAngle));
	circumferencePoint.z = circleCenterPoint.z + circleRadius * (unitVectorU.z * cos(arcAngle) + unitVectorV.z * sin(arcAngle));
	return circumferencePoint;
}

//���������߽��� ���������������ⷢ��㣩
__device__ Point GetIntersectionOfTwoRays(Ray ray1,Ray ray2)
{
	  float a1=ray1.direction.x,a2=ray1.direction.y,a3=ray1.direction.z;
	  float b1=-ray2.direction.x,b2=-ray2.direction.y,b3=-ray2.direction.z;
	  float c1=ray2.originalPoint.x-ray1.originalPoint.x,c2=ray2.originalPoint.y-ray1.originalPoint.y,c3=ray2.originalPoint.z-ray1.originalPoint.z;
	  float det=a1*(b2*c3-b3*c2)-b1*(a2*c3-a3*c2)+c1*(a2*b3-a3*b2);
	  Point intersection;
	  if(abs(det)<0.00001)
	  {
		  if (c1*b2-c2*b1<0.00001)
		  {
			  intersection.x=ray1.originalPoint.x;
			  intersection.y=ray1.originalPoint.y;
			  intersection.z=ray1.originalPoint.z;

		  }
		  else if ((abs(b2*a1-b1*a2)>0.00001))
		  {

			  float u=(c1*b2-c2*b1)/(b2*a1-b1*a2);
			  //float v=(c2*a1-c1*a2)/(b2*a1-b1*a2);
			  intersection.x=ray1.originalPoint.x+u*a1;
			  intersection.y=ray1.originalPoint.y+u*a2;
			  intersection.z=ray1.originalPoint.z+u*a3;
		  }
	  }
	  else
	  {
		  intersection.x=0;intersection.y=0;intersection.z=0;
	  }
	  return intersection;
}

//��ĳһ���ǶȻ��ֻ������Բ�ϵĵ�
__device__ Point *GetcircumPointOfTheCircle(Edge sameEdge, Point circleCenterPoint, float circleRadius, 
	Point unitCircleVectorU, Point unitCircleVectorV, float circleRange, int circumPointCount)
{
	float angleOfTwoCircumPoint = circleRange * PI /(circumPointCount * 180);//ȡ��ĽǶ�
	struct Point *circumPoints=new struct Point[circumPointCount];
	//��һ����ȡһ��΢С�Ƕȣ����������غ�
	circumPoints[0] = GetPointInCircumference(circleCenterPoint, circleRadius, unitCircleVectorU, unitCircleVectorV, 0.01);
	//��Բ��ȡ�㣬���붯̬����
	for (int i = 1; i < circumPointCount-1; i++)
	{
		circumPoints[i] = GetPointInCircumference(circleCenterPoint, circleRadius, unitCircleVectorU, unitCircleVectorV, i * angleOfTwoCircumPoint);
	}
	//���һ�������һ��΢С�Ƕȣ����������غ�
	circumPoints[circumPointCount-1] = GetPointInCircumference(circleCenterPoint, circleRadius, 
		unitCircleVectorU, unitCircleVectorV, circleRange * PI/180-0.01);
	return circumPoints;
}

//�����������ߵĺ˺��� diffractionEdge�������p1�������p2֮��Ĳ��� diffractionRayCount��ʾÿ������������������ߵ�����
__global__ void GenerateDiffractionRays(TriangleRayTube *incidentRayTubes, Edge *diffractionEdge, Face *diffractionFace1, Face *diffractionFace2, 
	Ray *diffractionRays1, Ray *diffractionRays2, QuadrangleRayTube *diffractionRayTubes, unsigned const int diffractionRayCount)
{
	//�ú˺����У�diffractionEdge��������Ĺ��ܣ�ֻ������������㡣 ֻ�к˺����ľֲ�����sameEdge����������Ĺ���
	int idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	
	Point originPoint = GetIntersectionOfTwoRays(incidentRayTubes[idx].ray1, incidentRayTubes[idx].ray2);//�������ⷢ���
	Ray inRay1=CreateNewInRay(originPoint, diffractionEdge[idx].p1);//�����ⷢ��㵽�����1���¹���һ��������
	Ray inRay2=CreateNewInRay(originPoint, diffractionEdge[idx].p2);//�����ⷢ��㵽�����2���¹���һ��������
	float angleOfRay1AndEdge=GetAngleOfTwoStraightLines(inRay1.direction, GetDirectionVectorOfEdge(diffractionEdge[idx]));//��ȡ����1�����ߵļн�
	float angleOfRay2AndEdge=GetAngleOfTwoStraightLines(inRay2.direction, GetDirectionVectorOfEdge(diffractionEdge[idx]));//��ȡ����2�����ߵļн�
	Edge sameEdge=GetEdgeOfAdjacentFaces(diffractionFace1[idx],diffractionFace2[idx]);//���ߵ������⣬diffractionEdgeΪ���Ӽ�
	Point circleCenterPoint1 = GetCircleCenterPoint(angleOfRay1AndEdge, inRay1, sameEdge, diffractionEdge[idx].p1);//��������ѡһ������������ͬ���Ҳ��������ĵ�,һ��ѡ�������ĳ������,���н�Ϊ90��ʱ����������ΪԲ�̣�Բ��Ϊ�����
	Point circleCenterPoint2 = GetCircleCenterPoint(angleOfRay2AndEdge, inRay2, sameEdge, diffractionEdge[idx].p2);
	float circleRadius1 = GetCircleRadius(angleOfRay1AndEdge, diffractionEdge[idx].p1, circleCenterPoint1);//����Բ1�İ뾶
	float circleRadius2 = GetCircleRadius(angleOfRay2AndEdge, diffractionEdge[idx].p2, circleCenterPoint2);//����Բ2�İ뾶
	Point circleVectorU1 = GetVectorInThePlaneOfCircle(sameEdge, diffractionFace1[idx]);//����Բ����ƽ���ϵ�һ������
	Point circleVectorU2 = circleVectorU1;//���һ����������diffractionFace1����Ϊֻ��Ҫ��������һ��������
	Point circleVectorV1 = GetAnotherVectorInThePlaneOfCircle(sameEdge, circleVectorU1);//����һ����ǰ����������������������������������������Ҳ��Բ���ڵ�ƽ����
	Point circleVectorV2 = GetAnotherVectorInThePlaneOfCircle(sameEdge, circleVectorU2);
	//������Բ������������Ľ���
	Point triPoint1;
	Point triPoint2;
	triPoint1 = GetRemainPointOnFace(diffractionFace1[idx], sameEdge);
	triPoint2 = GetRemainPointOnFace(diffractionFace2[idx], sameEdge);
	Point crossPointOfFace1AndCircle1 = GetCrossPointOfCircleWithTer(diffractionFace1[idx], circleCenterPoint1, 
		circleRadius1, circleVectorU1, circleVectorV1, triPoint1);//Բ1��������1һ��Ľ���
	Point crossPointOfFace2AndCircle1 = GetCrossPointOfCircleWithTer(diffractionFace2[idx], circleCenterPoint1, 
		circleRadius1, circleVectorU1, circleVectorV1, triPoint2);//Բ1��������2һ��Ľ���
	Point crossPointOfFace1AndCircle2 = GetCrossPointOfCircleWithTer(diffractionFace1[idx], circleCenterPoint2, 
		circleRadius2, circleVectorU2, circleVectorV2, triPoint1);//Բ2��������1һ��Ľ���
	Point crossPointOfFace2AndCircle2 = GetCrossPointOfCircleWithTer(diffractionFace2[idx], circleCenterPoint2, 
		circleRadius2, circleVectorU2, circleVectorV2, triPoint2);//Բ2��������2һ��Ľ���
	Point circlePointVector1;
	Point circlePointVector2;
	if(IsTwoPointSame(circleCenterPoint1, diffractionEdge[idx].p1))//�����������Բ�ģ���ȡ������Ϊ��������
	{
		circlePointVector1 = C_UnitizeVectorOnGPU(GetDirectionVectorOfEdge(sameEdge));
	}
	else//�������㲻����Բ�ģ�����������Ϊ��������
	{
		circlePointVector1.x = circleCenterPoint1.x - diffractionEdge[idx].p1.x;
		circlePointVector1.y = circleCenterPoint1.y - diffractionEdge[idx].p1.y;
		circlePointVector1.z = circleCenterPoint1.z - diffractionEdge[idx].p1.z;
		circlePointVector1 = C_UnitizeVectorOnGPU(circlePointVector1);
	}
	if(IsTwoPointSame(circleCenterPoint2, diffractionEdge[idx].p2))//�����������Բ�ģ���ȡ������Ϊ��������
	{
		circlePointVector2 = C_UnitizeVectorOnGPU(GetDirectionVectorOfEdge(sameEdge));
	}
	else//�������㲻����Բ�ģ�����������Ϊ��������
	{
		circlePointVector2.x = circleCenterPoint2.x - diffractionEdge[idx].p2.x;
		circlePointVector2.y = circleCenterPoint2.y - diffractionEdge[idx].p2.y;
		circlePointVector2.z = circleCenterPoint2.z - diffractionEdge[idx].p2.z;
		circlePointVector2 = C_UnitizeVectorOnGPU(circlePointVector2);
	}
	SetUnitVectorVnCirclePlane(circleVectorU1, circleVectorV1, circleCenterPoint1, 
		crossPointOfFace1AndCircle1, crossPointOfFace2AndCircle1, circlePointVector1, 90);//�����µ�����ϵ
	SetUnitVectorVnCirclePlane(circleVectorU2, circleVectorV2, circleCenterPoint2, 
		crossPointOfFace1AndCircle2, crossPointOfFace2AndCircle2, circlePointVector1, 90);
	struct Point* circumPoints1 = new struct Point[diffractionRayCount];//�����1��Ӧ��Բ���ϵĵ�
	struct Point* circumPoints2 = new struct Point[diffractionRayCount];//�����2��Ӧ��Բ���ϵĵ�
	circumPoints1 = GetcircumPointOfTheCircle(sameEdge, circleCenterPoint1, circleRadius1, circleVectorU1, circleVectorV1, 270, diffractionRayCount);
	circumPoints2 = GetcircumPointOfTheCircle(sameEdge, circleCenterPoint2, circleRadius2, circleVectorU2, circleVectorV2, 270, diffractionRayCount);
    //������Ȧ�������� ÿ��thread�е�ÿ������㣨�������㣩����diffractionRayCount���������ߣ���������㹲����2*diffractionRayCount���������ߣ���������Ԥ�����������
	unsigned int j = 0;//���ڱ�ʾԲ���ϵĵ������circumPoints���
	for(unsigned int i = idx * diffractionRayCount; i < (idx+1) * diffractionRayCount; i++)
	{
		diffractionRays1[i].originalPoint = diffractionEdge[idx].p1;//�����1
		diffractionRays2[i].originalPoint = diffractionEdge[idx].p2;//�����2
		diffractionRays1[i].direction.x = circumPoints1[j].x - diffractionRays1[i].originalPoint.x;//�����1����������һ����������
        diffractionRays1[i].direction.y = circumPoints1[j].y - diffractionRays1[i].originalPoint.y;
		diffractionRays1[i].direction.z = circumPoints1[j].z - diffractionRays1[i].originalPoint.z;
		diffractionRays2[i].direction.x = circumPoints2[j].x - diffractionRays2[i].originalPoint.x;//�����2����������һ����������
        diffractionRays2[i].direction.y = circumPoints2[j].y - diffractionRays2[i].originalPoint.y;
		diffractionRays2[i].direction.z = circumPoints2[j].z - diffractionRays2[i].originalPoint.z;
		j++;
	}

	//�������߹�
	unsigned int k = idx * diffractionRayCount;
	for(unsigned int i = idx * (diffractionRayCount-1); i < (idx+1) * (diffractionRayCount - 1); i++)
	{
		//�������ǽ��ܹ���
		diffractionRayTubes[i].ray1 = diffractionRays1[k];
		diffractionRayTubes[i].ray2 = diffractionRays1[k+1];
		diffractionRayTubes[i].ray3 = diffractionRays2[k+1];
		diffractionRayTubes[i].ray4 = diffractionRays2[k];
		//����Ϊ·���ڵ�ļ�¼
		if(incidentRayTubes[idx].path.nodeLevel==0)
		{
			diffractionRayTubes[i].path.nodeLevel = 1;
			diffractionRayTubes[i].path.node1.point1 = sameEdge.p1;
			diffractionRayTubes[i].path.node1.point2 = sameEdge.p2;
		}
		if(incidentRayTubes[idx].path.nodeLevel==1)
		{
			diffractionRayTubes[i].path.node1 = incidentRayTubes[idx].path.node1;
			diffractionRayTubes[i].path.nodeLevel = 2;
			diffractionRayTubes[i].path.node2.point1 = sameEdge.p1;
			diffractionRayTubes[i].path.node2.point2 = sameEdge.p2;
		}
		if(incidentRayTubes[idx].path.nodeLevel==2)
		{
			diffractionRayTubes[i].path.node1 = incidentRayTubes[idx].path.node1;
			diffractionRayTubes[i].path.node2 = incidentRayTubes[idx].path.node2;
			diffractionRayTubes[i].path.nodeLevel = 3;
			diffractionRayTubes[i].path.node3.point1 = sameEdge.p1;
			diffractionRayTubes[i].path.node3.point2 = sameEdge.p2;
		}
		k++;
	}
	delete [] circumPoints1;
	delete [] circumPoints2;
	//������begin
	/*f1[idx]=circleRadius1;
	f2[idx]=circleRadius2;
	Point U1=crossPointOfFace1AndCircle1;
	Point U2=crossPointOfFace2AndCircle1;
	test1[idx].x=U1.x;
	test1[idx].y=U1.y;
	test1[idx].z=U1.z;
	test2[idx].x=U2.x;
	test2[idx].y=U2.y;
	test2[idx].z=U2.z;*/
	//������end
}

//diffractionRayCount��ʾÿ������������������ߵ����� RaysToBeDiffractedCount��ʾ�������ߵ�������incidentRayTubes�ĸ�����
hipError_t GetDiffractionRaysInGPU(TriangleRayTube *incidentRayTubes, Edge *diffractionEdge, Face *diffractionFace1, Face *diffractionFace2, 
	Ray *diffractionRays1, Ray *diffractionRays2, QuadrangleRayTube *diffractionRayTubes,
	unsigned const int diffractionRayCount, unsigned const int RaysToBeDiffractedCount)
{
	hipError_t cudaStatus;
	TriangleRayTube *d_incidentRayTubes = 0;
	Edge *d_diffractionEdge = 0;
	Face *d_diffractionFace1 = 0;
	Face *d_diffractionFace2 = 0;
	Ray *d_diffractionRays1 = 0;
	Ray *d_diffractionRays2 = 0;
	QuadrangleRayTube *d_diffractionRayTubes = 0;
	/*Point *d_test1;
	Point *d_test2;
	float *d_f1;
	float *d_f2;*/

	cudaStatus=hipSetDevice(0);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"CUDA capable GPU is not available!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_incidentRayTubes, RaysToBeDiffractedCount*sizeof(TriangleRayTube));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_incidentRayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionEdge, RaysToBeDiffractedCount*sizeof(Edge));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionEdge hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionFace1, RaysToBeDiffractedCount*sizeof(Face));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionFace1 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionFace2, RaysToBeDiffractedCount*sizeof(Face));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionFace2 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionRays1, diffractionRayCount*RaysToBeDiffractedCount*sizeof(Ray));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionRays1 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionRays2, diffractionRayCount*RaysToBeDiffractedCount*sizeof(Ray));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionRays2 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_diffractionRayTubes, (diffractionRayCount-1)*RaysToBeDiffractedCount*sizeof(QuadrangleRayTube));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_diffractionRayTubes hipMalloc error!");
		goto Error;
	}

	//������begin
	/*cudaStatus = hipMalloc((void**)&d_test1, RaysToBeDiffractedCount*sizeof(Point));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_test1 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_test2, RaysToBeDiffractedCount*sizeof(Point));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_test2 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_f1, RaysToBeDiffractedCount*sizeof(float));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_f1 hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_f2, RaysToBeDiffractedCount*sizeof(float));
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr, "d_f2 hipMalloc error!");
		goto Error;
	}*/
	//������end
	cudaStatus = hipMemcpy(d_incidentRayTubes, incidentRayTubes, RaysToBeDiffractedCount*sizeof(TriangleRayTube), hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"incidentRayTubes Memcpy failed!");
	}
	cudaStatus = hipMemcpy(d_diffractionEdge, diffractionEdge, RaysToBeDiffractedCount*sizeof(Edge), hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionEdge Memcpy failed!");
	}
	cudaStatus = hipMemcpy(d_diffractionFace1, diffractionFace1, RaysToBeDiffractedCount*sizeof(Face), hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionFace1 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(d_diffractionFace2, diffractionFace2, RaysToBeDiffractedCount*sizeof(Face), hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionFace2 Memcpy failed!");
	}

	const int num_blocks=1;
	const int num_threads=RaysToBeDiffractedCount;
	GenerateDiffractionRays<<<num_blocks, num_threads>>>(d_incidentRayTubes, d_diffractionEdge, d_diffractionFace1, d_diffractionFace2, 
	d_diffractionRays1, d_diffractionRays2, d_diffractionRayTubes, diffractionRayCount);
	cudaStatus = hipMemcpy(diffractionRays1, d_diffractionRays1, diffractionRayCount*RaysToBeDiffractedCount*sizeof(Ray),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionRays1 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(diffractionRays2, d_diffractionRays2, diffractionRayCount*RaysToBeDiffractedCount*sizeof(Ray),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionRays2 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(diffractionRayTubes, d_diffractionRayTubes, (diffractionRayCount-1)*RaysToBeDiffractedCount*sizeof(QuadrangleRayTube),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionRayTubes Memcpy failed!");
	}
	//������begin
	/*cudaStatus = hipMemcpy(test1, d_test1, RaysToBeDiffractedCount*sizeof(Point),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"test1 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(test2, d_test2, RaysToBeDiffractedCount*sizeof(Point),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"test2 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(f1, d_f1, RaysToBeDiffractedCount*sizeof(float),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"f1 Memcpy failed!");
	}
	cudaStatus = hipMemcpy(f2, d_f2, RaysToBeDiffractedCount*sizeof(float),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"f1 Memcpy failed!");
	}*/
	//������end
	return cudaStatus;
Error:
	hipFree(d_incidentRayTubes);
	hipFree(d_diffractionEdge);
	hipFree(d_diffractionFace1);
	hipFree(d_diffractionFace2);
	hipFree(d_diffractionRays1);
	hipFree(d_diffractionRays2);
	hipFree(d_diffractionRayTubes);
	/*hipFree(d_test1);
	hipFree(d_test2);
	hipFree(d_f1);
	hipFree(d_f2);*/
}