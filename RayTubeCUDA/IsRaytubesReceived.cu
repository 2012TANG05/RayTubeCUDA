#include "hip/hip_runtime.h"
#include "IsRaytubesReceived.cuh"
//���������߹��Ƿ���ڽ��ջ���



__device__ bool IRR_IsTriangleRayTubesReceived(TriangleRayTube triangRayTube, Point Receiver)
{
	return IRR_IsRayTubesReceived(triangRayTube.ray1, triangRayTube.ray2, triangRayTube.ray3, Receiver, 0);
}


//�ı������߹��Ƿ���ڽ��ջ���
__device__ bool IRR_IsTQuadrangleRayTubesReceived(QuadrangleRayTube QuadrangleRayTube, Point Receiver)
{
	int RaytubeReceived1 = IsDiffractionRayTubesReceivedOnGPU(QuadrangleRayTube.ray1, QuadrangleRayTube.ray2, QuadrangleRayTube.ray3, Receiver);
	int RaytubeReceived2 = IsDiffractionRayTubesReceivedOnGPU(QuadrangleRayTube.ray2, QuadrangleRayTube.ray3, QuadrangleRayTube.ray4, Receiver);
	int RaytubeReceived3 = IsDiffractionRayTubesReceivedOnGPU(QuadrangleRayTube.ray1, QuadrangleRayTube.ray2, QuadrangleRayTube.ray4, Receiver);
	int RaytubeReceived4 = IsDiffractionRayTubesReceivedOnGPU(QuadrangleRayTube.ray1, QuadrangleRayTube.ray3, QuadrangleRayTube.ray4, Receiver);
	if(RaytubeReceived1 + RaytubeReceived2 + RaytubeReceived3 + RaytubeReceived4 == 0)
		return false;
	return true;
} 
//�����ж��������ߣ�һ�����������߹ܣ��Ƿ���ڽ��յ���
__device__ bool  IRR_IsRayTubesReceived(Ray ray1, Ray ray2, Ray ray3, Point receive, int flag)
{
	Point virtualOriginalPoint = CGetIntersectionOfTwoRaysOnGPU(ray1, ray2); 
	float x = receive.x - virtualOriginalPoint.x;
	float y = receive.y - virtualOriginalPoint.y;
	float z = receive.z - virtualOriginalPoint.z;
	double Denominator = ray1.direction.x * ray2.direction.y * ray3.direction.z 
		- ray1.direction.x * ray3.direction.y * ray2.direction.z + ray2.direction.x 
		* ray3.direction.y * ray1.direction.z - ray2.direction.x * ray1.direction.y * ray3.direction.z
		 + ray3.direction.x * ray1.direction.y * ray2.direction.z - ray3.direction.x * ray2.direction.y * ray1.direction.z;
	double uNumerator = x * ray2.direction.y * ray3.direction.z - x * ray3.direction.y * ray2.direction.z
		+ ray2.direction.x * ray3.direction.y * z - ray2.direction.x * y * ray3.direction.z
		+ ray3.direction.x * y * ray2.direction.z - ray3.direction.x * ray2.direction.y * z;
	double vNumerator = ray1.direction.x * y * ray3.direction.z - ray1.direction.x * ray3.direction.y * z
		+ x * ray3.direction.y * ray1.direction.z - x * ray1.direction.y * ray3.direction.z
		+ ray3.direction.x * ray1.direction.y * z - ray3.direction.x * y * ray1.direction.z;
	double tNumerator = ray1.direction.x * ray2.direction.y * z - ray1.direction.x * y * ray2.direction.z
		+ ray2.direction.x * y * ray1.direction.z - ray2.direction.x * ray1.direction.y * z
		+ x * ray1.direction.y * ray2.direction.z - x * ray2.direction.y * ray1.direction.z;


	if(abs(Denominator)>0.000001)
	{
		double u = uNumerator / Denominator;
		double v = vNumerator / Denominator;
		double t = tNumerator / Denominator;
		if(flag == 0)
		{	
			if(u >0 && v > 0 && t>0)
				return true;
		}
		else if(flag == 1)
		{
			if(u>0 && u<1 && v>0 && v<1 && t>0 && t<1)
				return true;
		}
	}
	return false;

}

//�������߹��жϽ��յĺ˺���
__global__ void CalculateIsTriangleRayTubesReceivedOnGPU(TriangleRayTube *rayTubes, 
	Point receive, bool *identifier,int rayTubeCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<rayTubeCount)
	{	identifier[i] = IRR_IsTriangleRayTubesReceived(rayTubes[i], receive);
		/*if(identifier[i] == true)
			printf("%d\n",i);*/
	}
}
//�ı������߹��ж��Ƿ���յĺ˺���
__global__ void CalculateIsQuadrangleRayTubesReceivedOnGPU(QuadrangleRayTube *rayTubes, 
	Point receive, bool *identifier, int rayTubeCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<rayTubeCount)
	identifier[i] = IRR_IsTQuadrangleRayTubesReceived(rayTubes[i], receive);
}

//��CPU�е����ݴ���GPU��
hipError_t GetTriangleRayTubesReceived(TriangleRayTube *rayTubes,bool *identifier, Point receive, int rayTubeCount)
{
	TriangleRayTube *device_rayTubes;
	bool *device_identifier;
//	Point device_receive;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&device_rayTubes, rayTubeCount * sizeof(TriangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, rayTubeCount *sizeof(bool));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_identifier hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_rayTubes, rayTubes, rayTubeCount * sizeof(TriangleRayTube), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "rayTubes Memcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_identifier, identifier, rayTubeCount * sizeof(bool), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy failed!\n");
		goto Error;
	}
	const int num_blocks = 32;
	const int num_threads = 640;
	CalculateIsTriangleRayTubesReceivedOnGPU<<<num_blocks, num_threads>>>(device_rayTubes, receive, device_identifier, rayTubeCount);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }
	cudaStatus = hipMemcpy(identifier, device_identifier, rayTubeCount * sizeof(bool), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy Error!\n");
		goto Error;
	}


	
	Error:
		hipFree(device_rayTubes);
		hipFree(device_identifier);

	return cudaStatus;
}



hipError_t GetQuadrangleRayTubesReceived(QuadrangleRayTube *rayTubes,bool *identifier, Point receive, int rayTubeCount)
{
	QuadrangleRayTube *device_rayTubes;
	bool *device_identifier;
//	Point device_receive;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&device_rayTubes, rayTubeCount * sizeof(QuadrangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, rayTubeCount *sizeof(int));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_identifier hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_rayTubes, rayTubes, rayTubeCount * sizeof(QuadrangleRayTube), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "rayTubes Memcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_identifier, identifier, rayTubeCount * sizeof(bool), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy failed!\n");
		goto Error;
	}
	const int num_blocks = 32;
	const int num_threads = 640;
	CalculateIsQuadrangleRayTubesReceivedOnGPU<<<num_blocks, num_threads>>>(device_rayTubes, receive, device_identifier, rayTubeCount);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }
	cudaStatus = hipMemcpy(identifier, device_identifier, rayTubeCount * sizeof(bool), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy Error!\n");
		goto Error;
	}


	
	Error:
		hipFree(device_rayTubes);
		hipFree(device_identifier);

	return cudaStatus;
}

//�ж��������߹��Ƿ���ڴ��ڽ��ջ���
__device__ bool IsDiffractionRayTubesReceivedOnGPU(Ray ray1, Ray ray2, Ray ray3, Point reciever)
{
	Point originalPoint;
	Ray temRay1, temRay2, temRay3, temRay4, temRay5, temRay6;
	if(ray1.originalPoint.x == ray2.originalPoint.x && ray1.originalPoint.y == ray2.originalPoint.y && ray1.originalPoint.z == ray2.originalPoint.z)
	{
		originalPoint=ray1.originalPoint;
		GetNewRayOnGPU(&temRay1, &temRay2, &temRay3, &temRay4, ray1, ray2, ray3);
		if(CalculateIsDiffractionRecievedOnGPU(temRay1, temRay2, temRay3, reciever, originalPoint) == true)
			return true;
		else
			return IRR_IsRayTubesReceived(ray1, ray2, temRay4, reciever, 1);

	}
	else if(ray1.originalPoint.x == ray3.originalPoint.x && ray1.originalPoint.y == ray3.originalPoint.y && ray1.originalPoint.z == ray3.originalPoint.z)
	{
		originalPoint=ray1.originalPoint;
		GetNewRayOnGPU(&temRay1, &temRay3, &temRay2, &temRay4, ray1, ray3, ray2);
		if(CalculateIsDiffractionRecievedOnGPU(temRay3, temRay1, temRay2, reciever, originalPoint) == true)
			return true;
		else
			return IRR_IsRayTubesReceived(ray1, ray3, temRay4, reciever, 1);
	}
	else if(ray3.originalPoint.x == ray2.originalPoint.x && ray3.originalPoint.y == ray2.originalPoint.y && ray3.originalPoint.z == ray2.originalPoint.z)
	{
		originalPoint=ray2.originalPoint;
		GetNewRayOnGPU(&temRay3, &temRay2, &temRay1, &temRay4, ray3, ray2, ray1);
		if(CalculateIsDiffractionRecievedOnGPU(temRay3, temRay2, temRay1, reciever, originalPoint) == true)
			return true;
		else
			return IRR_IsRayTubesReceived(ray3, ray2, temRay4, reciever, 1);
	}
	else
	{
		printf("error:�������߹�û���ҵ�����\n");
		return false;
	}
}


//�õ��µ����߹�
__device__ void GetNewRayOnGPU(Ray *temRay1, Ray *temRay2, Ray *temRay3, Ray *temRay4, Ray ray1, Ray ray2, Ray ray3)
{
	(*temRay1).originalPoint.x = ray1.originalPoint.x + ray1.direction.x;
	(*temRay1).originalPoint.y = ray1.originalPoint.y + ray1.direction.y;
	(*temRay1).originalPoint.z = ray1.originalPoint.z + ray1.direction.z;
	(*temRay2).originalPoint.x = ray2.originalPoint.x + ray2.direction.x;
	(*temRay2).originalPoint.y = ray2.originalPoint.y + ray2.direction.y;
	(*temRay2).originalPoint.z = ray2.originalPoint.z + ray2.direction.z;
	(*temRay3).originalPoint = ray3.originalPoint;
	(*temRay1).direction = ray1.direction;
	(*temRay2).direction = ray2.direction;
	(*temRay3).direction = ray3.direction;
	(*temRay4).originalPoint = ray1.originalPoint;
	(*temRay4).direction.x = ray3.originalPoint.x - ray1.originalPoint.x;
	(*temRay4).direction.y = ray3.originalPoint.y - ray1.originalPoint.y;
	(*temRay4).direction.z = ray3.originalPoint.z - ray1.originalPoint.z;
}

//�����������߹��Ƿ���յ�
__device__ bool CalculateIsDiffractionRecievedOnGPU(Ray temRay1, Ray temRay2, Ray temRay3, Point reciever,Point originalPoint)
{
	Point intersection1, intersection2 ,intersection3;
	float x1 = temRay1.originalPoint.x, x2 = temRay2.originalPoint.x, x3 = temRay3.originalPoint.x;
	float y1 = temRay1.originalPoint.y, y2 = temRay2.originalPoint.y, y3 = temRay3.originalPoint.y;
	float z1 = temRay1.originalPoint.z, z2 = temRay2.originalPoint.z, z3 = temRay3.originalPoint.z;
	float A = (y2 - y1) * (z3 - z1) - (z2 - z1) * (y3 - y1);
	float B = (x3 - x1) * (z2 - z1) - (x2 - x1) * (z3 - z1);
	float C = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
	float D = -(A * reciever.x + B * reciever.y + C * reciever.z);
	if(JudgeStraightAndFaceOnGPU(A, B, C, D, temRay1, originalPoint, &intersection1) == true 
		&&JudgeStraightAndFaceOnGPU(A, B, C, D, temRay2, originalPoint, &intersection2) == true 
		&&JudgeStraightAndFaceOnGPU(A, B, C, D, temRay3, originalPoint, &intersection3) == true)
	{
		float x0 = reciever.x - intersection1.x, x1 = intersection2.x - intersection1.x, x2 = intersection3.x - intersection1.x;
		float y0 = reciever.y - intersection1.y, y1 = intersection2.y - intersection1.y, y2 = intersection3.y - intersection1.y;
		float z0 = reciever.z - intersection1.z, z1 = intersection2.z - intersection1.z, z2 = intersection3.z - intersection1.z;
		float denominator = x1 * y2 - x2 * y1;
		if(abs(denominator) < 0.000001)
			return false;
		float u_numerator = x0 * y2 - x2 * y0;
		float v_numerator = x1 * y0 - x0 * y1;
		float u = u_numerator / denominator;
		float v = v_numerator / denominator;
		if(u>0 && u<1 && v>0 && v<1)
			return true;
	}
	return false;
}

//�ж�ֱ�������Ƿ��н��㲢����
__device__ bool JudgeStraightAndFaceOnGPU(float A, float B, float C, float D, Ray ray, Point originalPoint, Point *intersection)
{
	float x0 = ray.originalPoint.x, x1 = ray.direction.x;
	float y0 = ray.originalPoint.y, y1 = ray.direction.y;
	float z0 = ray.originalPoint.z, z1 = ray.direction.z;
	float denominator = A * x1 * x1 + B * x1 * y1 + C * x1 * z1;
	if(abs(denominator) < 0.000001)
		return false;
	float x_numerator = -D * x1 * x1 + B * x1 * (x1 * y0 - y1 * x0) + C * x1 * (x1 * z0 - z1 * x0);
	float x = x_numerator / denominator;
	if((x - x0) / x1 < 0)
		return false;
	float y_numerator = -A * x1 * (x1 * y0 - y1 * x0) + D * y1 * (x1 * z0 - z1 * x0) + C * y1 * (x1 * z0 - z1 * x0) - C * z1 * (x1 * y0 - y1 * x0);
	float z_numerator = -A * x1 * (x1 * z0 - z1 * x0) + B * z1 * (x1 * y0 - y1 * x0) - B * y1 * (x1 * z0 - z1 * x0) - D * x1 * z1;
	float y = y_numerator / denominator;
	float z = z_numerator / denominator;
	(*intersection).x = x;
	(*intersection).y = y;
	(*intersection).z = z;
	return true;
}


__device__ Point CGetIntersectionOfTwoRaysOnGPU(Ray ray1,Ray ray2)
{
	  float a1=ray1.direction.x,a2=ray1.direction.y,a3=ray1.direction.z;
	  float b1=-ray2.direction.x,b2=-ray2.direction.y,b3=-ray2.direction.z;
	  float c1=ray2.originalPoint.x-ray1.originalPoint.x,c2=ray2.originalPoint.y-ray1.originalPoint.y,c3=ray2.originalPoint.z-ray1.originalPoint.z;
	  float det=a1*(b2*c3-b3*c2)-b1*(a2*c3-a3*c2)+c1*(a2*b3-a3*b2);
	  Point intersection;
	  intersection.x=0;intersection.y=0;intersection.z=0;
	  if(abs(det)<0.001)
	  {
		  if (abs(c1*b2-c2*b1)<0.001)
		  {
			  intersection.x=ray1.originalPoint.x;
			  intersection.y=ray1.originalPoint.y;
			  intersection.z=ray1.originalPoint.z;

		  }
		  else if ((abs(b2*a1-b1*a2)>0.00001))
		  {

			  float u=(c1*b2-c2*b1)/(b2*a1-b1*a2);
			  //float v=(c2*a1-c1*a2)/(b2*a1-b1*a2);
			  intersection.x=ray1.originalPoint.x+u*a1;
			  intersection.y=ray1.originalPoint.y+u*a2;
			  intersection.z=ray1.originalPoint.z+u*a3;
		  }
	  }
	  else
	  {
		  intersection.x=0;intersection.y=0;intersection.z=0;
	  }
	  return intersection;
}