#include "hip/hip_runtime.h"
#include "IsRaytubesReceivedOnGPU.cuh"
//���������߹��Ƿ���ڽ��ջ���


//���������߹��Ƿ���ڽ��ջ��ϣ����������߹��������)
__device__ bool IsTriangleRayTubesReceivedOnGPU(TriangleRayTube triangRayTube, Point receiver)
{
	return ChooseReceiveMethodByIntersectionOnGPU(triangRayTube.ray1, triangRayTube.ray2, triangRayTube.ray3, receiver);
}


//�ı������߹��Ƿ���ڽ��ջ��ϣ��ı������߹��жϽ�����ڣ�
__device__ bool IsQuadrangleRayTubesReceivedOnGPU(QuadrangleRayTube quadrangleRayTube, Point receiver)
{
	int RaytubeReceived1 = ChooseReceiveMethodByIntersectionOnGPU(quadrangleRayTube.ray1, quadrangleRayTube.ray2, quadrangleRayTube.ray3, receiver);
	int RaytubeReceived2 = ChooseReceiveMethodByIntersectionOnGPU(quadrangleRayTube.ray2, quadrangleRayTube.ray3, quadrangleRayTube.ray4, receiver);
	int RaytubeReceived3 = ChooseReceiveMethodByIntersectionOnGPU(quadrangleRayTube.ray1, quadrangleRayTube.ray2, quadrangleRayTube.ray4, receiver);
	int RaytubeReceived4 = ChooseReceiveMethodByIntersectionOnGPU(quadrangleRayTube.ray1, quadrangleRayTube.ray3, quadrangleRayTube.ray4, receiver);
	if(RaytubeReceived1 + RaytubeReceived2 + RaytubeReceived3 + RaytubeReceived4 == 0)
		return false;
	return true;
} 


__device__ bool ChooseReceiveMethodByIntersectionOnGPU(Ray ray1, Ray ray2, Ray ray3, Point receiver)
{
	Ray temRay1 = ray1, temRay2 = ray2, temRay3 = ray3, temRay4;
	int flag = 0;
	int judges = JudgeIntersectionOfThreeRayOnGPU(&temRay1, &temRay2, &temRay3);
	if(judges == 0)
	{
		return IsRayTubesWithoutSameStartPointReceivedOnGPU(temRay1, temRay2, temRay3, receiver);
	}
	else if(judges == 3)
	{
		return IsRayTubesWithSameStartPointReceivedOnGPU(temRay1, temRay2, temRay3, receiver, flag);		
	}
	else if(judges == 1)
	{
		while(judges == 1)//��ֹ����һ�������Ϸ��������������ǵ��������ߵ��������
		{
			flag = 1;
			GetNewRayTubeOnGPU(&temRay1, &temRay2, &temRay3, &temRay4, 1);//�����1Ӧ����flagÿ�����ӵ�ֵ���
			judges = JudgeIntersectionOfThreeRayOnGPU(&temRay1, &temRay2, &temRay3);
		}
		if(IsRayTubesWithoutSameStartPointReceivedOnGPU(temRay1, temRay2, temRay3, receiver) == true)
			return true;
		else
			return IsRayTubesWithSameStartPointReceivedOnGPU(temRay1, temRay2, temRay4, receiver, flag);
	}
	
}


//�ж����߹��ڵ������Ƿ��н��㲢���ؽ������
__device__ int JudgeIntersectionOfThreeRayOnGPU(Ray *ray1, Ray *ray2, Ray *ray3)
{
	if(JudgePointEqualOnGPU((*ray1).originalPoint, (*ray2).originalPoint) == true 
		&& JudgePointEqualOnGPU((*ray2).originalPoint, (*ray3).originalPoint) == true ) 
	{//������������һ��
		return 3;
	}
	else if(JudgePointEqualOnGPU((*ray1).originalPoint, (*ray2).originalPoint) == true)
	{//��һ��������һ��
		return 1;
	}
	else if(JudgePointEqualOnGPU((*ray1).originalPoint, (*ray3).originalPoint) == true)
	{//��һ��������һ�㣬��������������ڶ�������λ�ý��������ں�������
		ExchangeTwoRayOnGPU(ray2, ray3);
		return 1;
	}
	else if(JudgePointEqualOnGPU((*ray2).originalPoint, (*ray3).originalPoint) == true)
	{//�ڶ���������һ�㣬����һ�������ڵ���������λ�ý��������ں�������
		ExchangeTwoRayOnGPU(ray1, ray3);
		return 1;
	}
	else
		return 0;
}


//����õ��µ����߹�
__device__ void GetNewRayTubeOnGPU(Ray *temRay1, Ray *temRay2, Ray *temRay3, Ray *temRay4, int flag)
{
	(*temRay4).originalPoint = (*temRay1).originalPoint;
	(*temRay4).direction.x = (*temRay3).originalPoint.x - (*temRay1).originalPoint.x;
	(*temRay4).direction.y = (*temRay3).originalPoint.y - (*temRay1).originalPoint.y;
	(*temRay4).direction.z = (*temRay3).originalPoint.z - (*temRay1).originalPoint.z;
	(*temRay1).originalPoint.x = (*temRay1).originalPoint.x + (*temRay1).direction.x * flag;
	(*temRay1).originalPoint.y = (*temRay1).originalPoint.y + (*temRay1).direction.y * flag;
	(*temRay1).originalPoint.z = (*temRay1).originalPoint.z + (*temRay1).direction.z * flag;
	(*temRay2).originalPoint.x = (*temRay2).originalPoint.x + (*temRay2).direction.x * flag;
	(*temRay2).originalPoint.y = (*temRay2).originalPoint.y + (*temRay2).direction.y * flag;
	(*temRay2).originalPoint.z = (*temRay2).originalPoint.z + (*temRay2).direction.z * flag;
}


//û�й�ͬ�������߹ܵĽ���
__device__ bool IsRayTubesWithoutSameStartPointReceivedOnGPU(Ray temRay1, Ray temRay2, Ray temRay3, Point receiver)
{
	Point intersection1, intersection2 ,intersection3;
	float x1 = temRay1.originalPoint.x, x2 = temRay2.originalPoint.x, x3 = temRay3.originalPoint.x;
	float y1 = temRay1.originalPoint.y, y2 = temRay2.originalPoint.y, y3 = temRay3.originalPoint.y;
	float z1 = temRay1.originalPoint.z, z2 = temRay2.originalPoint.z, z3 = temRay3.originalPoint.z;
	float A = (y2 - y1) * (z3 - z1) - (z2 - z1) * (y3 - y1);
	float B = (x3 - x1) * (z2 - z1) - (x2 - x1) * (z3 - z1);
	float C = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
	float D = -(A * receiver.x + B * receiver.y + C * receiver.z);
	if(JudgeIntersectionOfStraightAndFaceOnGPU(A, B, C, D, temRay1, &intersection1) == true 
		&&JudgeIntersectionOfStraightAndFaceOnGPU(A, B, C, D, temRay2, &intersection2) == true 
		&&JudgeIntersectionOfStraightAndFaceOnGPU(A, B, C, D, temRay3, &intersection3) == true)
	{
		float x0 = receiver.x - intersection1.x, x1 = intersection2.x - intersection1.x, x2 = intersection3.x - intersection1.x;
		float y0 = receiver.y - intersection1.y, y1 = intersection2.y - intersection1.y, y2 = intersection3.y - intersection1.y;
		float z0 = receiver.z - intersection1.z, z1 = intersection2.z - intersection1.z, z2 = intersection3.z - intersection1.z;
		float denominator = x1 * y2 - x2 * y1;
		if(abs(denominator) < 0.000001)
			return false;
		float u_numerator = x0 * y2 - x2 * y0;
		float v_numerator = x1 * y0 - x0 * y1;
		float u = u_numerator / denominator;
		float v = v_numerator / denominator;
		if(u>=0 && u<=1 && v>=0 && v<=1 && u+v>=0 && u+v<=1)
			return true;

	}
	return false;

}


//�ж�ֱ�������Ƿ��н��㲢����
//��ֱ�߷��̺�ƽ�淽���������
__device__ bool JudgeIntersectionOfStraightAndFaceOnGPU(float A, float B, float C, float D, Ray ray, Point *intersection)
{
	float x0 = ray.originalPoint.x, x1 = ray.direction.x;
	float y0 = ray.originalPoint.y, y1 = ray.direction.y;
	float z0 = ray.originalPoint.z, z1 = ray.direction.z;
	float denominator = A * x1 * x1 + B * x1 * y1 + C * x1 * z1;
	if(abs(denominator) < 0.000001)
		return false;
	float x_numerator = -D * x1 * x1 - B * x1 * (x1 * y0 - y1 * x0) - C * x1 * (x1 * z0 - z1 * x0);
	float x = x_numerator / denominator;
	float y_numerator = A * x1 * (x1 * y0 - y1 * x0) - D * y1 * x1 - C * y1 * (x1 * z0 - z1 * x0) + C * z1 * (x1 * y0 - y1 * x0);
	float z_numerator = A * x1 * (x1 * z0 - z1 * x0) - B * z1 * (x1 * y0 - y1 * x0) + B * y1 * (x1 * z0 - z1 * x0) - D * x1 * z1;
	float y = y_numerator / denominator;
	float z = z_numerator / denominator;
	if((x - x0) / x1 < 0)
		return false;
	(*intersection).x = x;
	(*intersection).y = y;
	(*intersection).z = z;
	return true;
}


//���߹ܽ����жϣ����գ�
bool IsRayTubesWithSameStartPointReceivedOnGPU(Ray ray1, Ray ray2, Ray ray3, Point receiver, int flag)
{
	Point virtualOriginalPoint = ray1.originalPoint;
	float x = receiver.x - virtualOriginalPoint.x;
	float y = receiver.y - virtualOriginalPoint.y;
	float z = receiver.z - virtualOriginalPoint.z;
	double Denominator = ray1.direction.x * ray2.direction.y * ray3.direction.z 
		- ray1.direction.x * ray3.direction.y * ray2.direction.z + ray2.direction.x 
		* ray3.direction.y * ray1.direction.z - ray2.direction.x * ray1.direction.y * ray3.direction.z
		+ ray3.direction.x * ray1.direction.y * ray2.direction.z - ray3.direction.x * ray2.direction.y * ray1.direction.z;
	double uNumerator = x * ray2.direction.y * ray3.direction.z - x * ray3.direction.y * ray2.direction.z
		+ ray2.direction.x * ray3.direction.y * z - ray2.direction.x * y * ray3.direction.z
		+ ray3.direction.x * y * ray2.direction.z - ray3.direction.x * ray2.direction.y * z;
	double vNumerator = ray1.direction.x * y * ray3.direction.z - ray1.direction.x * ray3.direction.y * z
		+ x * ray3.direction.y * ray1.direction.z - x * ray1.direction.y * ray3.direction.z
		+ ray3.direction.x * ray1.direction.y * z - ray3.direction.x * y * ray1.direction.z;
	double tNumerator = ray1.direction.x * ray2.direction.y * z - ray1.direction.x * y * ray2.direction.z
		+ ray2.direction.x * y * ray1.direction.z - ray2.direction.x * ray1.direction.y * z
		+ x * ray1.direction.y * ray2.direction.z - x * ray2.direction.y * ray1.direction.z;


	if(abs(Denominator)>0.000001)
	{
		double u = uNumerator / Denominator;
		double v = vNumerator / Denominator;
		double t = tNumerator / Denominator;
		if(flag == 0)
		{	if(u >=0 && v >= 0 && t>=0)
				return true;
		}
		else
		{
			if(u>=0 && u<=flag && v>=0 && v<=flag && t>=0 && t<=flag)
				return true;
		}
		
	}
	return false;

}



//�������߹��жϽ��յĺ˺���
__global__ void CalculateIsTriangleRayTubesReceivedOnGPU(TriangleRayTube *rayTubes, 
	Point receiver, bool *identifier,int rayTubeCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<rayTubeCount)
	{	identifier[i] = IsTriangleRayTubesReceivedOnGPU(rayTubes[i], receiver);
		if(identifier[i] == true)
			printf("%d\n",i);
	}
}


//�ı������߹��ж��Ƿ���յĺ˺���
__global__ void CalculateIsQuadrangleRayTubesReceivedOnGPU(QuadrangleRayTube *rayTubes, 
	Point receiver, bool *identifier, int rayTubeCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<rayTubeCount)
	identifier[i] = IsQuadrangleRayTubesReceivedOnGPU(rayTubes[i], receiver);
	
}


//��CPU�е����ݴ���GPU��
hipError_t GetTriangleRayTubesReceived(TriangleRayTube *rayTubes,bool *identifier, Point receiver, int rayTubeCount)
{
	TriangleRayTube *device_rayTubes;
	bool *device_identifier;
//	Point device_receive;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&device_rayTubes, rayTubeCount * sizeof(TriangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, rayTubeCount *sizeof(bool));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_identifier hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_rayTubes, rayTubes, rayTubeCount * sizeof(TriangleRayTube), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "rayTubes Memcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_identifier, identifier, rayTubeCount * sizeof(bool), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy failed!\n");
		goto Error;
	}
	const int num_blocks = 32;
	const int num_threads = 640;
	CalculateIsTriangleRayTubesReceivedOnGPU<<<num_blocks, num_threads>>>(device_rayTubes, receiver, device_identifier, rayTubeCount);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }
	cudaStatus = hipMemcpy(identifier, device_identifier, rayTubeCount * sizeof(bool), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy Error!\n");
		goto Error;
	}


	
	Error:
		hipFree(device_rayTubes);
		hipFree(device_identifier);

	return cudaStatus;
}



hipError_t GetQuadrangleRayTubesReceived(QuadrangleRayTube *rayTubes,bool *identifier, Point receiver, int rayTubeCount)
{
	QuadrangleRayTube *device_rayTubes;
	bool *device_identifier;
//	Point device_receive;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&device_rayTubes, rayTubeCount * sizeof(QuadrangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, rayTubeCount *sizeof(int));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_identifier hipMalloc error!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_rayTubes, rayTubes, rayTubeCount * sizeof(QuadrangleRayTube), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "rayTubes Memcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_identifier, identifier, rayTubeCount * sizeof(bool), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy failed!\n");
		goto Error;
	}
	const int num_blocks = 32;
	const int num_threads = 640;
	CalculateIsQuadrangleRayTubesReceivedOnGPU<<<num_blocks, num_threads>>>(device_rayTubes, receiver, device_identifier, rayTubeCount);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }
	cudaStatus = hipMemcpy(identifier, device_identifier, rayTubeCount * sizeof(bool), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier Memcpy Error!\n");
		goto Error;
	}


	
	Error:
		hipFree(device_rayTubes);
		hipFree(device_identifier);

	return cudaStatus;
}



__device__ bool JudgePointEqualOnGPU(Point a, Point b)//�ж��������Ƿ����
{
	if(a.x == b.x && a.y == b.y && a.z == b.z)
		return true;
	return false;
}

__device__ void ExchangeTwoRayOnGPU(Ray *ray1, Ray *ray2)//������������
{
	Ray tempRay;
	tempRay = *ray1;
	*ray1 = *ray2;
	*ray2 = tempRay;
}