#include "hip/hip_runtime.h"
#include"IsRaytubesRecieved.cuh"




//���������߹��Ƿ���ڽ��ջ���
 bool IsTriangleRayTubesReceived(TriangleRayTube triangRayTube, Point Receiver)
{
	return IsRayTubesReceived(triangRayTube.ray1, triangRayTube.ray2, triangRayTube.ray3, Receiver);
}


//�ı������߹��Ƿ���ڽ��ջ���
 bool IsTQuadrangleRayTubesReceived(QuadrangleRayTube QuadrangleRayTube, Point Receiver)
{
	int Raytubereceive1 = IsRayTubesReceived(QuadrangleRayTube.ray1, QuadrangleRayTube.ray2, QuadrangleRayTube.ray3, Receiver);
	int Raytubereceive2 = IsRayTubesReceived(QuadrangleRayTube.ray2, QuadrangleRayTube.ray3, QuadrangleRayTube.ray4, Receiver);
	int Raytubereceive3 = IsRayTubesReceived(QuadrangleRayTube.ray1, QuadrangleRayTube.ray2, QuadrangleRayTube.ray4, Receiver);
	int Raytubereceive4 = IsRayTubesReceived(QuadrangleRayTube.ray1, QuadrangleRayTube.ray3, QuadrangleRayTube.ray4, Receiver);
	if(Raytubereceive1 + Raytubereceive2 + Raytubereceive3 + Raytubereceive4 == 0)
		return false;
	return true;
} 

bool  IsRayTubesReceived(Ray ray1, Ray ray2, Ray ray3, Point receive)
{
	 
	float x = receive.x - ray1.originalPoint.x - ray2.originalPoint.x - ray3.originalPoint.x;
	float y = receive.y - ray1.originalPoint.y - ray2.originalPoint.y - ray3.originalPoint.y;
	float z = receive.z - ray1.originalPoint.z - ray2.originalPoint.z - ray3.originalPoint.z;
	double Denominator = ray1.direction.x * ray2.direction.y * ray3.direction.z 
		- ray1.direction.x * ray3.direction.y * ray2.direction.z + ray2.direction.x 
		* ray3.direction.y * ray1.direction.z - ray2.direction.x * ray1.direction.y * ray3.direction.z
		 + ray3.direction.x * ray1.direction.y * ray2.direction.z - ray3.direction.x * ray2.direction.y * ray1.direction.z;
	double uNumerator = x * ray2.direction.y * ray3.direction.z - x * ray3.direction.y * ray2.direction.z
		+ ray2.direction.x * ray3.direction.y * z - ray2.direction.x * y * ray3.direction.z
		+ ray1.direction.x * y * ray3.direction.z - ray1.direction.x * ray2.direction.y * z;
	double vNumerator = ray1.direction.x * y * ray3.direction.z - ray1.direction.x * ray3.direction.y * z
		+ x * ray3.direction.y * ray1.direction.z - x * ray1.direction.y * ray3.direction.z
		+ ray3.direction.x * ray1.direction.y * z - ray1.direction.x * y * ray3.direction.z;
	double tNumerator = ray1.direction.x * ray2.direction.y * z - ray1.direction.x * y * ray2.direction.z
		+ ray2.direction.x * y * ray1.direction.z - ray2.direction.x * ray1.direction.y * z
		+ x * ray1.direction.y * ray2.direction.z - x * ray2.direction.y * ray3.direction.z;


	if(abs(Denominator)>0.01)
	{
		double u = uNumerator / Denominator;
		double v = vNumerator / Denominator;
		double t = tNumerator / Denominator;
		if(u >0 && v > 0 && t>0)
			return true;
	}

	return false;

}



