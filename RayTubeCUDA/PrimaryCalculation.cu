#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "Face.h"
#include "BasicTool.h"

using namespace std;
#define PI 3.1415926;

 __device__ void PC_UnitizeVector(float *a,float *b,float *c)//��λ������
{
	float length=sqrt((*a)*(*a)+(*b)*(*b)+(*c)*(*c));
	*a=*a/length;
	*b=*b/length;
	*c=*c/length;
}
 
 __device__ Vector PC_GetReflectedVector(Vector d,Vector n)//���㷴�����߷�������
{
	Vector reflectedVector;
	float temp=2*(d.x*n.x+d.y*n.y+d.z*n.z);
	reflectedVector.x=d.x-temp*n.x;
	reflectedVector.y=d.y-temp*n.y;
	reflectedVector.z=d.z-temp*n.z;
	return reflectedVector;
}


 //__global__ void CalculationReflectionAndDiffractionWithBuildingFace(TriangleRayTube *incidentRayTubes,TriangleRayTube *reflectedRayTubes,float *CFace,Edge *diffractionEdge,float *distance,int faceCount,int rayTubeCount)
 __global__ void CalculationReflectionAndDiffractionWithBuildingFace(int *a)
 {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	*a=i;

}



hipError_t  GetReflectionAndDiffractionWithBuildingFace(TriangleRayTube *originalRayTubes,TriangleRayTube *reflectedRayTubes,float *buildingFace,bool *isDiffractionExist,int buildingFaceCount,int rayTubeCount)
{
	const int pointPerFace=9;
	hipError_t cudaStatus;

	TriangleRayTube *device_incidentRayTubes=0;
	TriangleRayTube *device_reflectedRayTubes=0;
	Edge *device_edge=0;
	float *device_face=0;
	float *device_distance=0;
	int *a=0;


	cudaStatus=hipSetDevice(0);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"CUDA capable GPU is not available!");
		goto Error;
	}

	cudaStatus=hipMalloc((void**)&device_incidentRayTubes,rayTubeCount*sizeof(TriangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_incidentRayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_reflectedRayTubes,rayTubeCount*2*sizeof(TriangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_reflectedRayTubes hipMalloc error!");

		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_distance,rayTubeCount*3*sizeof(TriangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_distance hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_edge,rayTubeCount*sizeof(Edge));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_edge hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_face,buildingFaceCount*pointPerFace*sizeof(float));
	cudaStatus=hipMalloc((void**)&a,20480*sizeof(int));

	const int num_blocks=32;
	const int num_threads=640;
	CalculationReflectionAndDiffractionWithBuildingFace<<<num_blocks,num_threads>>>(a);
	return cudaStatus;
Error:
	;

	
}