#include "hip/hip_runtime.h"
#include "ReflectionWithBuildingCalculationOnGPU.cuh"

__device__ void RWBC_UitizeVectorOnGPU(float *a,float *b,float *c)//��λ������
{
	float length=sqrt((*a)*(*a)+(*b)*(*b)+(*c)*(*c));
	*a=*a/length;
	*b=*b/length;
	*c=*c/length; 
}

__device__ Point RWBC_GetReflectedVectorOnGPU(Point d,Point n)//���㷴�����߷�������
{
	Point reflectedPoint;
	float temp=2*(d.x*n.x+d.y*n.y+d.z*n.z);
	reflectedPoint.x=d.x-temp*n.x;
	reflectedPoint.y=d.y-temp*n.y;
	reflectedPoint.z=d.z-temp*n.z;
	return reflectedPoint;
}
__device__ Ray RWBC_CalculateReflectedRayOnGPU(Ray incidentRay,Face *face,int faceCount,int *reflectedFace,int *flag)
{
	Ray reflectedRay;
	float t=50000;
	for (int i=0;i<faceCount;i++)
	{
		float a1=-incidentRay.direction.x,a2=-incidentRay.direction.y,a3=-incidentRay.direction.z;
		float b1=face[i]. B.x-face[i].A.x,b2=face[i].B.y-face[i].A.y,b3=face[i].B.z-face[i].A.z;
		float c1=face[i]. C.x-face[i].A.x,c2=face[i].C.y-face[i].A.y,c3=face[i].C.z-face[i].A.z;
		float x1=incidentRay.originalPoint.x-face[i].A.x,x2=incidentRay.originalPoint.y-face[i].A.y,x3=incidentRay.originalPoint.z-face[i].A.z;
		float denominator=a1*(b2*c3-b3*c2)-b1*(a2*c3-a3*c2)+c1*(a2*b3-a3*b2);
		float t_numerator=x1*(b2*c3-b3*c2)-b1*(x2*c3-x3*c2)+c1*(x2*b3-x3*b2);
		float u_numerator=a1*(x2*c3-x3*c2)-x1*(a2*c3-a3*c2)+c1*(a2*c3-a3*c2);
		float v_numerator=a1*(b2*x3-b3*x2)-b1*(a2*x3-a3*x2)+x1*(a2*b3-a3*b2);
		if (abs(denominator)>0.000001)
		{
			float u=u_numerator/denominator;
			float v=v_numerator/denominator;
			if(t_numerator/denominator<t&&t_numerator/denominator>1)
			{
				t=t_numerator/denominator;
				reflectedRay.originalPoint.x=incidentRay.originalPoint.x+incidentRay.direction.x*t;
				reflectedRay.originalPoint.y=incidentRay.originalPoint.y+incidentRay.direction.y*t;
				reflectedRay.originalPoint.z=incidentRay.originalPoint.z+incidentRay.direction.z*t;
				Point n;
				n.x=b2*c3-b3*c2;n.y=b3*c1-b1*c3;n.z=b1*c2-c1*b2;
				RWBC_UitizeVectorOnGPU(&n.x,&n.y,&n.z);
				reflectedRay.direction=RWBC_GetReflectedVectorOnGPU(incidentRay.direction,n);
				if ((u>0)&&(u<1)&&((u+v)>0)&&((u+v)<1))
				{
					*flag=1;
					*reflectedFace=i;
				}
			}
		}
	}
	return reflectedRay;
}

__device__ bool RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(int faceNumber1,int faceNumber2)
{
	int a=faceNumber1%12;
	int b=faceNumber2%12;
	if (faceNumber1==faceNumber2)
	{
		return true;
	} 
	if (a%2==0)
	{
		if (b==a+1)
		{
			return  true;
		}
	}
	if (a%2==1)
	{
		if (b==a-1)
		{
			return true;
		}
	}
	return false;
}

__device__ bool RWBC_JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(int faceNumber1,int faceNumber2)
{
	int a=faceNumber1%12;
	int b=faceNumber2%12;
	if (a%2==0&&a!=0)
	{
		if (b==a-1)
		{
			return  true;
		}
	}
	if (a%2==1&&a!=7)
	{
		if (b==a+1)
		{
			return true;
		}
	}
	if ((a==0&&b==7)||(a==7&&b==0))
	{
		return true;
	}
	return false;
}


__device__ void RWBC_GetThreeRemainingNumbersOnGPU(int number,int *remainingNumber1,int *remainingNumber2,int *remainingNumber3)
{
	if (number==0)
	{
		*remainingNumber1=1;
		*remainingNumber2=2;
		*remainingNumber3=3;
	}
	if (number==1)
	{
		*remainingNumber1=0;
		*remainingNumber2=2;
		*remainingNumber3=3;
	}
	if (number==2)
	{
		*remainingNumber1=0;
		*remainingNumber2=1;
		*remainingNumber3=3;
	}
	if (number==3)
	{
		*remainingNumber1=0;
		*remainingNumber2=1;
		*remainingNumber3=2;
	}
}
__device__ void RWBC_GetOneRemainingNumberOnGPU(int number1,int number2,int number3,int *remainingNumber)
{
	*remainingNumber=6-number1-number2-number3;
}


__device__ void GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(int flagNumber1,int flagNumber2,int *reflectedFace,Ray *reflectedRays,Ray *incidentRays,QuadrangleRayTube *reflectedRayTubes,Face *face,int *reflectionExist,int i)
{
	if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[flagNumber1],reflectedFace[flagNumber2]))
	{
		reflectionExist[i]=1;
		reflectedRayTubes[2*i].ray1=reflectedRays[0];
		reflectedRayTubes[2*i].ray2=reflectedRays[1];
		reflectedRayTubes[2*i].ray3=reflectedRays[2];
		reflectedRayTubes[2*i].ray4=reflectedRays[3];
		reflectedRayTubes[2*i+1].ray1=reflectedRays[0];
		reflectedRayTubes[2*i+1].ray2=reflectedRays[1];
		reflectedRayTubes[2*i+1].ray3=reflectedRays[2];
		reflectedRayTubes[2*i+1].ray4=reflectedRays[3];

	}
	if (RWBC_JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(reflectedFace[flagNumber1],reflectedFace[flagNumber2]))
	{
		reflectionExist[i]=1;
		int remainingNumber1,remainingNumber2,remainingNumber3;
		RWBC_GetThreeRemainingNumbersOnGPU(flagNumber1,&remainingNumber1,&remainingNumber2,&remainingNumber3);
		reflectedRayTubes[2*i].ray1=reflectedRays[flagNumber1];
		reflectedRayTubes[2*i].ray2=reflectedRays[remainingNumber1];
		reflectedRayTubes[2*i].ray3=reflectedRays[remainingNumber2];
		reflectedRayTubes[2*i].ray4=reflectedRays[remainingNumber2];
		RWBC_GetThreeRemainingNumbersOnGPU(flagNumber2,&remainingNumber1,&remainingNumber2,&remainingNumber3);
		reflectedRayTubes[2*i+1].ray1=reflectedRays[flagNumber2];
		reflectedRayTubes[2*i+1].ray2=reflectedRays[remainingNumber1];
		reflectedRayTubes[2*i+1].ray3=reflectedRays[remainingNumber2];
		reflectedRayTubes[2*i+1].ray4=reflectedRays[remainingNumber3];

	}	
}

__device__ void GetReflectedRayTubesWhenThreeFlagsEqualOneOnGPU(int flagNumber1,int flagNumber2,int flagNumber3,int *reflectedFace,Ray *reflectedRays,Ray *incidentRays,QuadrangleRayTube *reflectedRayTubes,QuadrangleRayTube *incidentRayTubes,Face *face,int *reflectionExist,int i)
{
	//int remainingNumber;
	//RWBC_GetOneRemainingNumberOnGPU(flagNumber1,flagNumber2,flagNumber3,&remainingNumber);
	if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[flagNumber1],reflectedFace[flagNumber2]))
	{
		RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[flagNumber1]],face[reflectedFace[flagNumber3]]);
		GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(flagNumber1,flagNumber3,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
	}
	if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[flagNumber1],reflectedFace[flagNumber3]))
	{
		RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[flagNumber2]],face[reflectedFace[flagNumber3]]);
		GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(flagNumber2,flagNumber3,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
	}
	if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[flagNumber2],reflectedFace[flagNumber3]))
	{
		RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[flagNumber1]],face[reflectedFace[flagNumber2]]);
		GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(flagNumber1,flagNumber2,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
	}
}

__device__ void RWBC_AddPathNodeToReflectedRayTubeOnGPU(QuadrangleRayTube *incidentRayTubes,QuadrangleRayTube *reflectedRayTubes,int i,Face face1,Face face2)
{
	if(incidentRayTubes[i].path.nodeLevel=0)
	{
		reflectedRayTubes[2*i].path.nodeLevel=1;
		reflectedRayTubes[2*i].path.node1.point1=face1.A;
		reflectedRayTubes[2*i].path.node1.point2=face1.B;
		reflectedRayTubes[2*i].path.node1.point3=face1.C;
		reflectedRayTubes[2*i+1].path.nodeLevel=1;
		reflectedRayTubes[2*i+1].path.node1.point1=face2.A;
		reflectedRayTubes[2*i+1].path.node1.point2=face2.B;
		reflectedRayTubes[2*i+1].path.node1.point3=face2.C;
	}
	if(incidentRayTubes[i].path.nodeLevel=1)
	{
		reflectedRayTubes[2*i].path.node1=incidentRayTubes[i].path.node1;
		reflectedRayTubes[2*i].path.nodeLevel=2;
		reflectedRayTubes[2*i].path.node2.point1=face1.A;
		reflectedRayTubes[2*i].path.node2.point2=face1.B;
		reflectedRayTubes[2*i].path.node2.point3=face1.C;
		reflectedRayTubes[2*i+1].path.nodeLevel=2;
		reflectedRayTubes[2*i+1].path.node1=incidentRayTubes[i].path.node1;
		reflectedRayTubes[2*i+1].path.node2.point1=face2.A;
		reflectedRayTubes[2*i+1].path.node2.point2=face2.B;
		reflectedRayTubes[2*i+1].path.node2.point3=face2.C;
	}
	if(incidentRayTubes[i].path.nodeLevel=2)
	{
		reflectedRayTubes[2*i].path.node1=incidentRayTubes[i].path.node1;
		reflectedRayTubes[2*i].path.node2=incidentRayTubes[i].path.node2;
		reflectedRayTubes[2*i].path.nodeLevel=3;
		reflectedRayTubes[2*i].path.node3.point1=face1.A;
		reflectedRayTubes[2*i].path.node3.point2=face1.B;
		reflectedRayTubes[2*i].path.node3.point3=face1.C;
		reflectedRayTubes[2*i+1].path.nodeLevel=3;
		reflectedRayTubes[2*i+1].path.node1=incidentRayTubes[i].path.node1;
		reflectedRayTubes[2*i+1].path.node2=incidentRayTubes[i].path.node2;
		reflectedRayTubes[2*i+1].path.node3.point1=face2.A;
		reflectedRayTubes[2*i+1].path.node3.point2=face2.B;
		reflectedRayTubes[2*i+1].path.node3.point3=face2.C;
	}
}

__global__ void CalculateReflectionWithBuildingFaceOnGPU(QuadrangleRayTube *incidentRayTubes,int faceCount,int rayTubeCount,QuadrangleRayTube *reflectedRayTubes,Face *face,int *reflectionExist)
{
	int i=blockIdx.x * blockDim.x+threadIdx.x;
	int reflectedFace[4];
	int flag[4]={0,0,0,0};
	Ray reflectedRays[4],incidentRays[4];
	incidentRays[0]=incidentRayTubes[i].ray1;
	incidentRays[1]=incidentRayTubes[i].ray2;
	incidentRays[2]=incidentRayTubes[i].ray3;
	incidentRays[3]=incidentRayTubes[i].ray4;
	reflectedRays[0]=RWBC_CalculateReflectedRayOnGPU(incidentRays[0],face,faceCount,&reflectedFace[0],&flag[0]);
	reflectedRays[1]=RWBC_CalculateReflectedRayOnGPU(incidentRays[1],face,faceCount,&reflectedFace[1],&flag[1]);
	reflectedRays[2]=RWBC_CalculateReflectedRayOnGPU(incidentRays[2],face,faceCount,&reflectedFace[2],&flag[2]);
	reflectedRays[3]=RWBC_CalculateReflectedRayOnGPU(incidentRays[3],face,faceCount,&reflectedFace[3],&flag[3]);
	if (flag[0]+flag[1]+flag[2]+flag[3]==0)
	{
		reflectionExist[i]=0;
	}
	if (flag[0]+flag[1]+flag[2]+flag[3]==1)
	{
		reflectionExist[i]=1;
		//reflectedRayTubes[2*i].path.nodeLevel=1;
		reflectedRayTubes[2*i].ray1=reflectedRays[0];
		reflectedRayTubes[2*i].ray2=reflectedRays[1];
		reflectedRayTubes[2*i].ray3=reflectedRays[2];
		reflectedRayTubes[2*i].ray4=reflectedRays[3];
		//reflectedRayTubes[2*i+1].path.nodeLevel=1;
		reflectedRayTubes[2*i+1].ray1=reflectedRays[0];
		reflectedRayTubes[2*i+1].ray2=reflectedRays[1];
		reflectedRayTubes[2*i+1].ray3=reflectedRays[2];
		reflectedRayTubes[2*i+1].ray4=reflectedRays[3];
		for (int j=0;j<4;j++)
		{
			if (flag[j]==1)
			{
				RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[j]],face[reflectedFace[j]]);
			}
		}
	}
	if (flag[0]+flag[1]+flag[2]+flag[3]==2)
	{
		int m,n;
		for (int j=0;j<4;j++)
		{
			if (flag[j]==1)
			{
				m=j;
				for (int t=j+1;t<4;t++)
				{
					if(flag[t]==1)
					{
						n=t;
					}
				}
			}
		}
		GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(m,n,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
		RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[m]],face[reflectedFace[n]]);
	}
	if (flag[0]+flag[1]+flag[2]+flag[3]==3)
	{
		int j=0;
		int remainingNumber1,remainingNumber2,remainingNumber3;
		for (;j<4;j++)
		{
			if (flag[j]==0)
			{
				break;
			}
		}
		RWBC_GetThreeRemainingNumbersOnGPU(j,&remainingNumber1,&remainingNumber2,&remainingNumber3);
		GetReflectedRayTubesWhenThreeFlagsEqualOneOnGPU(remainingNumber1,remainingNumber2,remainingNumber3,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,incidentRayTubes,face,reflectionExist,i);
		
	}
	if (flag[0]+flag[1]+flag[2]+flag[3]==4)
	{
		if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1]))
		{
			if (RWBC_JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(reflectedFace[0],reflectedFace[2]))
			{
				RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[0]],face[reflectedFace[2]]);
				GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(0,2,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
			}
			else
			{
				RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[1]],face[reflectedFace[3]]);
				GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(1,3,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
			}
		}
		if (RWBC_JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[3]))
		{
			if (RWBC_JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(reflectedFace[0],reflectedFace[1]))
			{
				RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[0]],face[reflectedFace[1]]);
				GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(0,1,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
			}
			else
			{
				RWBC_AddPathNodeToReflectedRayTubeOnGPU(incidentRayTubes,reflectedRayTubes,i,face[reflectedFace[2]],face[reflectedFace[3]]);
				GetReflectedRayTubesWhenTwoFlagsEqualOneOnGPU(2,3,reflectedFace,reflectedRays,incidentRays,reflectedRayTubes,face,reflectionExist,i);
			}
		}
	}
}


hipError_t  GetReflectionWithBuildingFaceOnGPU(QuadrangleRayTube *incidentRayTubes,int faceCount,int rayTubeCount,Face *buildingFace,QuadrangleRayTube *reflectedRayTubes,int *reflectionExist)
{
	const int pointPerFace=9;
	hipError_t cudaStatus;

	QuadrangleRayTube *device_incidentRayTubes=0;
	QuadrangleRayTube *device_reflectedRayTubes=0;

	Face *device_face=0;
	//float *device_distance=0;
	//int *device_faceNumber1=0;
	//int *device_faceNumber2=0;

	int *device_reflectionExist=0;
	//float *device_a=0;
	cudaStatus=hipSetDevice(0);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"CUDA capable GPU is not available!");
		goto Error;
	}

	cudaStatus=hipMalloc((void**)&device_incidentRayTubes,rayTubeCount*sizeof(QuadrangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_incidentRayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_reflectedRayTubes,rayTubeCount*2*sizeof(QuadrangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_reflectedRayTubes hipMalloc error!");

		goto Error;
	}

	cudaStatus=hipMalloc((void**)&device_face,faceCount*sizeof(Face));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_face hipMalloc error!");
		goto Error;
	}
	/*cudaStatus=hipMalloc((void**)&device_faceNumber1,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_faceNumber1 hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_faceNumber2,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_faceNumber2 hipMalloc error!");
		goto Error;
	}*/

	cudaStatus=hipMalloc((void**)&device_reflectionExist,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_reflectionExis hipMalloc error!");
		goto Error;
	}
	//cudaStatus=hipMalloc((void**)&device_a,rayTubeCount*sizeof(float));
	//if (cudaStatus!=hipSuccess)
	//{
	//	fprintf(stderr,"device_reflectionExis hipMalloc error!");
	//	goto Error;
	//}

	cudaStatus=hipMemcpy(device_incidentRayTubes,incidentRayTubes,rayTubeCount*sizeof(QuadrangleRayTube),hipMemcpyHostToDevice);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"incidentRayTubes Memcpy failed!");
	}
	cudaStatus=hipMemcpy(device_face,buildingFace,faceCount*sizeof(Face),hipMemcpyHostToDevice);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"face Memcpy failed!");
	}

	const int num_blocks=32;
	const int num_threads=640;

	//CalculateReflectionAndDiffractionWithBuildingFace<<<num_blocks,num_threads>>>(device_incidentRayTubes,faceCount,rayTubeCount,device_reflectedRayTubes,device_face,device_edge,device_faceNumber1,device_faceNumber2,device_reflectionExist,device_diffractionExist);

	cudaStatus=hipMemcpy(reflectedRayTubes,device_reflectedRayTubes,rayTubeCount*2*sizeof(QuadrangleRayTube),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"reflectedRayTubes hipMemcpy failed!");
	}
	//cudaStatus=hipMemcpy(faceNumber1,device_faceNumber1,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	//if (cudaStatus!=hipSuccess)
	//{
	//	fprintf(stderr,"faceNumber1 hipMemcpy failed!");
	//}
	//cudaStatus=hipMemcpy(faceNumber2,device_faceNumber2,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	//if (cudaStatus!=hipSuccess)
	//{
	//	fprintf(stderr,"faceNumber2 hipMemcpy failed!");
	//}

	cudaStatus=hipMemcpy(reflectionExist,device_reflectionExist,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"reflectionExist hipMemcpy failed!");
	}

	return cudaStatus;
Error:
	hipFree(device_incidentRayTubes);
	hipFree(device_reflectedRayTubes);
	hipFree(device_face);
	//hipFree(device_faceNumber1);
	//hipFree(device_faceNumber2);
	hipFree(device_reflectionExist);
}