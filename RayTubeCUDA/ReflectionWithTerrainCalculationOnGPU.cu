#include "hip/hip_runtime.h"
#include "ReflectionWithTerrainCalculationOnGPU.cuh"

//��λ������
 __device__ void RWTC_UnitizeVectorOnGPU(float *a, float *b, float *c)
{
	float length = sqrt((*a) * (*a) + (*b) * (*b) + (*c) * (*c));
	*a = *a / length;
	*b = *b / length;
	*c = *c / length;
}




//���㷴�����߷�������
__device__ Point GetReflectedVectorOnGPU(Point d,Point n)
{
	Point reflectedVector;
	float temp = 2 * (d.x * n.x + d.y * n.y + d.z * n.z);
	reflectedVector.x = d.x - temp * n.x;
	reflectedVector.y = d.y - temp * n.y;
	reflectedVector.z = d.z - temp * n.z;
	return reflectedVector;
}


//GPU�м��㷴�����߹�
//face��    ��������Face *
//incidentrayTubes�������߹�    ��������TriangleRayTube *
//ReflectedRayTubes�������߹�    ��������TriangleRayTube *
//faceCountPerBlockÿ��block�����Ŀ�����ݹ�������    ��������int
//identifier �ж��Ƿ��з������߹�ֻ��1��0    ��������int *
//temDistance �жϾ����С�����ҵ�����ķ���㣨Զ�Ĳ��ᱻ�򵽣�   ��������float*
__global__ void CalculateTempTriangleReflectedRayTubesWithTerrainOnGPU(Face *face, 
	TriangleRayTube *incidentrayTubes, TriangleRayTube *tempReflectedRayTubes, 
	int faceCountPerBlock, int *identifier, float *tempDistance)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int identifier1=0, identifier2=0, identifier3=0;
	Ray tempRay1, tempRay2, tempRay3;
	float tempDistance1, tempDistance2, tempDistance3;	
	int reflectedFace1 = -1, reflectedFace2 = -1, reflectedFace3 = -1;
	int identifierNumber = 3 * i;

	tempRay1 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray1, face, faceCountPerBlock, &reflectedFace1, &identifier1);
	JudgeAndChangeTempReflectedRay(&tempReflectedRayTubes[i].ray1, tempRay1, tempDistance, tempDistance1, identifier, identifier1, i, identifierNumber);
	tempRay2 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray2, face, faceCountPerBlock, &reflectedFace2, &identifier2);
	JudgeAndChangeTempReflectedRay(&tempReflectedRayTubes[i].ray2, tempRay2, tempDistance, tempDistance2, identifier, identifier2, i, identifierNumber+1);
	tempRay3 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray3, face, faceCountPerBlock, &reflectedFace3, &identifier3);
	JudgeAndChangeTempReflectedRay(&tempReflectedRayTubes[i].ray3, tempRay3, tempDistance, tempDistance3, identifier, identifier3, i, identifierNumber+2);
	//if(identifier[identifierNumber] == 0)
	//{
	//	tempReflectedRayTubes[i].ray1 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray1, face, faceCountPerBlock, &reflectedFace1, &identifier1);
	//	identifier[identifierNumber] = identifier1;
	//}
	//if(identifier[identifierNumber+1] == 0)
	//{
	//	tempReflectedRayTubes[i].ray2 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray2, face, faceCountPerBlock, &reflectedFace2, &identifier2);
	//	identifier[identifierNumber+1] = identifier2;
	//}
	//if(identifier[identifierNumber+2] == 0)
	//{
	//	tempReflectedRayTubes[i].ray3 = CalculateReflectedRayWithTerrainOnGPU(incidentrayTubes[i].ray3, face, faceCountPerBlock, &reflectedFace3, &identifier3);
	//	identifier[identifierNumber+2] = identifier3;
	//}
}



//����ָ��ԭ��ɽ����Ż�
//�жϲ��ı䷴������
__device__ void JudgeAndChangeTempReflectedRay(Ray *ray, Ray tempRay, float *Distance, float tempDistance, int *identifier, int tempIdentifier, int rayTubeNumber, int identifierNumber)
{
	if(tempIdentifier == 1)//��ʾ���յ��˷�������
	{
		if(Distance[identifierNumber] == 0)//��ʾ֮ǰû�н��յ���������
		{
			Distance[identifierNumber] = tempDistance;
			*ray = tempRay;
			identifier[identifierNumber] = tempIdentifier;
		}
		else if(Distance[identifierNumber] < tempDistance)
		{
			Distance[identifierNumber] = tempDistance;
			*ray = tempRay;
		}
	}	
}


//��CPU�е����ݴ���GPU�м���,����������Ľ������GPU
hipError_t GetTriangleReflectedRayTubes(TriangleRayTube *incidentRayTubes,TriangleRayTube *tempReflectedRayTubes,
	Face *face,unsigned int rayTubeCount,unsigned int faceCount,int *identifier,
	float *tempDistance)
{
	Face *device_face = 0;
	float *device_tempDistance = 0;
	int *device_identifier = 0;
	TriangleRayTube *device_incidentRayTubes = 0;
	TriangleRayTube *device_tempReflectedRayTubes = 0;
	hipError_t cudaStatus; 
	cudaStatus = hipMalloc((void**)&device_incidentRayTubes, rayTubeCount * sizeof(TriangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_tempReflectedRayTubes, rayTubeCount * sizeof(TriangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_face, faceCount * sizeof(Face));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "face hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, 3 * rayTubeCount * sizeof(int));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_tempDistance, 3 * rayTubeCount * sizeof(float));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempDistance hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_face, face, faceCount * sizeof(Face), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "face Memcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(device_incidentRayTubes, incidentRayTubes, rayTubeCount * sizeof(TriangleRayTube), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr,"incidentRayTubes Memcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(device_identifier, identifier, 3 * rayTubeCount * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr,"identifier Memcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(device_tempReflectedRayTubes, tempReflectedRayTubes, rayTubeCount * sizeof(TriangleRayTube), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr,"tempReflectedRayTubes Memcpy failed!");
        goto Error;
    }
	const int num_blocks = 32;
	const int num_threads = 640;
	CalculateTempTriangleReflectedRayTubesWithTerrainOnGPU<<<num_blocks, num_threads>>>(device_face, device_incidentRayTubes, device_tempReflectedRayTubes, faceCount, device_identifier, device_tempDistance);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(identifier, device_identifier, 3 * rayTubeCount * sizeof(int), hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "isRefletedRaysExist hipMemcpy error!");
		goto Error;
	}

	cudaStatus = hipMemcpy(tempReflectedRayTubes, device_tempReflectedRayTubes, rayTubeCount * sizeof(TriangleRayTube), hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempTriangleReflectedRayTubes hipMemcpy error!");
		goto Error;
	}

	cudaStatus = hipMemcpy(tempDistance,device_tempDistance, 3 * rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempDistance hipMemcpy error!");
		goto Error;
	}

Error:
	hipFree(device_incidentRayTubes);
	hipFree(device_tempReflectedRayTubes);
	hipFree(device_face);
	hipFree(device_identifier);
	hipFree(device_tempDistance);
	return cudaStatus;
}

int CalculateTriangleReflectedRayTubesWithTerrainOnGPU(const unsigned int rayTubeCount, 
	unsigned int faceCount, Face *p_face, TriangleRayTube *incidentRayTubes, 
	TriangleRayTube *ReflectedRayTubes, int *isReflectedRayTubesExist, float *distance)
{
//rayTubeCount���߹���Ŀ    ��������const unsigned int
//faceCount�����Ŀ    ��������unsigned int
//p_face������ָ��    ��������Face *
//incidentRayTubes�������߹�    ��������RayTube *
//ReflectedRayTubes�������߹�    ��������RayTube *
//isReflectedRayTubesExist�������߹��Ƿ����    ��������int *
//distance��������Ĳ��Ƿ�������    ��������float *
	static TriangleRayTube tempRayTubes[20480];
	static int identifier[61440];
	static float tempDistance[61440];
	//TriangleRayTube tempRayTubes[20480];
	//int identifier[20480];
	//float tempDistance[61440];

	memset(identifier, 0, 61440*sizeof(int));
	const int faceCountPerTime = 100;

	int time = faceCount / faceCountPerTime;
	int remainder = faceCount % faceCountPerTime;
	hipError_t cudaStatus;
	for(int i = 0; i <= time; i++)
	{
		if(i < time)
		{
			cudaStatus = GetTriangleReflectedRayTubes(incidentRayTubes, tempRayTubes, p_face, rayTubeCount, faceCountPerTime, identifier, tempDistance);

		}
		else
		{
			cudaStatus = GetTriangleReflectedRayTubes(incidentRayTubes, tempRayTubes, p_face, rayTubeCount, remainder, identifier, tempDistance);						
		}

		p_face += faceCountPerTime;		
	}
	for(int j = 0; j < rayTubeCount; j++)
	{
		isReflectedRayTubesExist[j] = identifier[3*j] * identifier[3*j+1] * identifier[3*j+2];
		if(isReflectedRayTubesExist[j] == 1)
		{				
			ReflectedRayTubes[j] = tempRayTubes[j];					
		}
	}
	return 0;
}






__device__ Ray CalculateReflectedRayWithTerrainOnGPU(Ray incidentRay,Face *face,int faceCount,int *reflectedFace,int *flag)
{
	Ray reflectedRay;
	Point defaultPoint;
	defaultPoint.x=0;defaultPoint.y=0;defaultPoint.z=0;
	reflectedRay.originalPoint=defaultPoint;
	reflectedRay.direction=defaultPoint;
	float t=50000;
 	for (int i=0;i<faceCount;i++)
	{
		float a1=-incidentRay.direction.x,a2=-incidentRay.direction.y,a3=-incidentRay.direction.z;
		float b1=face[i]. B.x-face[i].A.x,b2=face[i].B.y-face[i].A.y,b3=face[i].B.z-face[i].A.z;
		float c1=face[i]. C.x-face[i].A.x,c2=face[i].C.y-face[i].A.y,c3=face[i].C.z-face[i].A.z;
		float x1=incidentRay.originalPoint.x-face[i].A.x,x2=incidentRay.originalPoint.y-face[i].A.y,x3=incidentRay.originalPoint.z-face[i].A.z;
		float denominator=a1*(b2*c3-b3*c2)-b1*(a2*c3-a3*c2)+c1*(a2*b3-a3*b2);
		float t_numerator=x1*(b2*c3-b3*c2)-b1*(x2*c3-x3*c2)+c1*(x2*b3-x3*b2);
		float u_numerator=a1*(x2*c3-x3*c2)-x1*(a2*c3-a3*c2)+c1*(a2*x3-a3*x2);
		float v_numerator=a1*(b2*x3-b3*x2)-b1*(a2*x3-a3*x2)+x1*(a2*b3-a3*b2);
		if (abs(denominator)>0.000001)
		{
			float u=u_numerator/denominator;
			float v=v_numerator/denominator;
 			if(t_numerator/denominator<t&&t_numerator/denominator>1&&u>=0&&u<=1&&v>0&&(u+v)<1&&v<=1)
			{
				*flag=1;
				*reflectedFace=i;
				t=t_numerator/denominator;
				reflectedRay.originalPoint.x=u*b1+v*c1+face[i].A.x;
				reflectedRay.originalPoint.y=u*b2+v*c2+face[i].A.y;
				reflectedRay.originalPoint.z=u*b3+v*c3+face[i].A.z;
				Point n;
				n.x=b2*c3-b3*c2;n.y=b3*c1-b1*c3;n.z=b1*c2-c1*b2;
				RWTC_UnitizeVectorOnGPU(&n.x,&n.y,&n.z);
				reflectedRay.direction=GetReflectedVectorOnGPU(incidentRay.direction,n);
			}
		}
	}
	return reflectedRay;
}




















































__global__ void CalculateTempQuadrangleReflectedRayTubesOnGPU(Face *face, 
	QuadrangleRayTube *incidentrayTubes, QuadrangleRayTube *tempReflectedRayTubes, 
	int faceCountPerBlock, int *identifier, float *tempDistance)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int identifier1, identifier2, identifier3,identifier4;
	int distanceNumber = 4 * i;
//	RWTC_CalculateTempReflectedRays(face, incidentrayTubes[i].ray1, &tempReflectedRayTubes[i].ray1 , faceCountPerBlock, &identifier1, tempDistance, distanceNumber);
//	RWTC_CalculateTempReflectedRays(face, incidentrayTubes[i].ray2, &tempReflectedRayTubes[i].ray2, faceCountPerBlock, &identifier2, tempDistance, distanceNumber+1);
//	RWTC_CalculateTempReflectedRays(face, incidentrayTubes[i].ray3, &tempReflectedRayTubes[i].ray3, faceCountPerBlock, &identifier3, tempDistance, distanceNumber+2);
//	RWTC_CalculateTempReflectedRays(face, incidentrayTubes[i].ray3, &tempReflectedRayTubes[i].ray4, faceCountPerBlock, &identifier4, tempDistance, distanceNumber+3);
	identifier[i] = identifier1 * identifier2 * identifier3 * identifier4;
}


hipError_t GetQuadrangleReflectedRayTubes(QuadrangleRayTube *incidentRayTubes,QuadrangleRayTube *tempReflectedRayTubes,
	Face *face,unsigned int rayTubeCount,unsigned int faceCount,int *identifier,
	float *tempDistance)
{

	Face *device_face = 0;
	float *device_tempDistance = 0;
	int *device_identifier = 0;
	QuadrangleRayTube *device_incidentRayTubes = 0;
	QuadrangleRayTube *device_tempReflectedRayTubes = 0;
	hipError_t cudaStatus; 
	cudaStatus = hipMalloc((void**)&device_incidentRayTubes, rayTubeCount * sizeof(QuadrangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_tempReflectedRayTubes, rayTubeCount * sizeof(QuadrangleRayTube));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "device_rayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_face, faceCount * sizeof(Face));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "face hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_identifier, rayTubeCount * sizeof(int));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "identifier hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_tempDistance, 4 * rayTubeCount * sizeof(float));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempDistance hipMalloc error!");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_face, face, faceCount * sizeof(Face), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "face Memcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(device_incidentRayTubes, incidentRayTubes, rayTubeCount * sizeof(QuadrangleRayTube), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr,"incidentRayTubes Memcpy failed!");
        goto Error;
    }
	//dim3 DimGrid(5,15);
	//dim3 DimBlock(24,36);
	//CaculatetempReflectedRays<<<DimGrid,DimBlock>>>(device_face,device_incidentRays,device_tempReflectedRays,faceNumber);
	const int num_blocks = 32;
	const int num_threads = 640;




	CalculateTempQuadrangleReflectedRayTubesOnGPU<<<num_blocks, num_threads>>>(device_face, device_incidentRayTubes, device_tempReflectedRayTubes, faceCount, device_identifier, device_tempDistance);



	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error!", cudaStatus);
        goto Error;
    }
	cudaStatus = hipMemcpy(tempReflectedRayTubes, device_tempReflectedRayTubes, rayTubeCount * sizeof(QuadrangleRayTube), hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempQuadrangleRayTubeReflectedRayTubes hipMemcpy error!");
		goto Error;
	}
	cudaStatus = hipMemcpy(identifier, device_identifier, rayTubeCount * sizeof(int), hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "isRefletedRaysExist hipMemcpy error!");
		goto Error;
	}
	cudaStatus = hipMemcpy(tempDistance,device_tempDistance, 3 * rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);     
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "tempDistance hipMemcpy error!");
		goto Error;
	}
Error:
	hipFree(device_incidentRayTubes);
	hipFree(device_tempReflectedRayTubes);
	hipFree(device_face);
	hipFree(device_identifier);
	hipFree(device_tempDistance);
	return cudaStatus;
}




int CalculateQuadrangleReflectedRayTubes(const unsigned int rayTubeCount, 
	unsigned int faceCount, Face *p_face, QuadrangleRayTube *incidentRayTubes, 
	QuadrangleRayTube *ReflectedRayTubes, int *isReflectedRayTubesExist, float *distance)
{
//rayTubeCount���߹���Ŀ    ��������const unsigned int
//faceCount�����Ŀ    ��������unsigned int
//p_face������ָ��    ��������Face *
//incidentRayTubes�������߹�    ��������RayTube *
//ReflectedRayTubes�������߹�    ��������RayTube *
//isReflectedRayTubesExist�������߹��Ƿ����    ��������int *
//distance��������Ĳ��Ƿ�������    ��������float *
	static QuadrangleRayTube tempRayTubes[20480];
	static int identifier[20480];
	static float tempDistance[81920];
	//TriangleRayTube tempRayTubes[20480];
	//int identifier[20480];
	//float tempDistance[61440];


	const int faceCountPerTime = 3000;

	int time = faceCount / faceCountPerTime;
	int remainder = faceCount % faceCountPerTime;
	hipError_t cudaStatus;

	for(int i = 0; i <= time; i++)
	{
		if(i < time)
		{
			cudaStatus = GetQuadrangleReflectedRayTubes(incidentRayTubes, tempRayTubes, p_face, rayTubeCount, faceCountPerTime, identifier, tempDistance);
		}
		else
		{
			cudaStatus = GetQuadrangleReflectedRayTubes(incidentRayTubes, tempRayTubes, p_face, rayTubeCount, remainder, identifier, tempDistance);
		}
		for(int j = 0; j < rayTubeCount; j++)
		{
			int distanceNumber = 4 * j;
			if(identifier[j] == 1 && tempDistance[distanceNumber] < distance[distanceNumber])
			{				
				ReflectedRayTubes[j] = tempRayTubes[j];
				distance[distanceNumber] = tempDistance[distanceNumber];
				isReflectedRayTubesExist[j] = 1;					
			}
			if(identifier[j] == 1 && tempDistance[++distanceNumber] < distance[distanceNumber])
			{				
				ReflectedRayTubes[j] = tempRayTubes[j];
				distance[distanceNumber] = tempDistance[distanceNumber];
				isReflectedRayTubesExist[j] = 1;					
			}
			if(identifier[j] == 1 && tempDistance[++distanceNumber] < distance[distanceNumber])
			{				
				ReflectedRayTubes[j] = tempRayTubes[j];
				distance[distanceNumber] = tempDistance[distanceNumber];
				isReflectedRayTubesExist[j] = 1;					
			}

		}
		p_face += faceCountPerTime;		
	}
	return 0;
}