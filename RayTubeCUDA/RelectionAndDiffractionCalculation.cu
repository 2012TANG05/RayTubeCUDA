#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "Face.h"
#include "BasicTool.h"

using namespace std;
#define PI 3.1415926;

 __device__ void RADC_UitizePoint(float *a,float *b,float *c)//��λ������
{
	float length=sqrt((*a)*(*a)+(*b)*(*b)+(*c)*(*c));
	*a=*a/length;
	*b=*b/length;
	*c=*c/length;
}
 
 __device__ Point RADC_GetReflectedPoint(Point d,Point n)//���㷴�����߷�������
{
	Point reflectedPoint;
	float temp=2*(d.x*n.x+d.y*n.y+d.z*n.z);
	reflectedPoint.x=d.x-temp*n.x;
	reflectedPoint.y=d.y-temp*n.y;
	reflectedPoint.z=d.z-temp*n.z;
	return reflectedPoint;
}
 __device__ Ray CalculateReflectedRay(Ray incidentRay,Face *face,int faceCount,int *reflectedFace,int *flag)
 {
	 Ray reflectedRay;
	 float t=50000;
	 for (int i=0;i<faceCount;i++)
	 {
		 float a1=-incidentRay.direction.x,a2=-incidentRay.direction.y,a3=-incidentRay.direction.z;
		 float b1=face[i]. B.x-face[i].A.x,b2=face[i].B.y-face[i].A.y,b3=face[i].B.z-face[i].A.z;
		 float c1=face[i]. C.x-face[i].A.x,c2=face[i].C.y-face[i].A.y,c3=face[i].C.z-face[i].A.z;
		 float x1=incidentRay.originalPoint.x-face[i].A.x,x2=incidentRay.originalPoint.y-face[i].A.y,x3=incidentRay.originalPoint.z-face[i].A.z;
		 float denominator=a1*(b2*c3-b3*c2)-b1*(a2*c3-a3*c2)+c1*(a2*b3-a3*b2);
		 float t_numerator=x1*(b2*c3-b3*c2)-b1*(x2*c3-x3*c2)+c1*(x2*b3-x3*b2);
		 float u_numerator=a1*(x2*c3-x3*c2)-x1*(a2*c3-a3*c2)+c1*(a2*c3-a3*c2);
		 float v_numerator=a1*(b2*x3-b3*x2)-b1*(a2*x3-a3*x2)+x1*(a2*b3-a3*b2);
		 if (abs(denominator)>0.000001)
		 {
			 float u=u_numerator/denominator;
			 float v=v_numerator/denominator;
			 if(t_numerator/denominator<t&&t_numerator/denominator>1)
			 {
				 t=t_numerator/denominator;
				 reflectedRay.originalPoint.x=incidentRay.originalPoint.x+incidentRay.direction.x*t;
				 reflectedRay.originalPoint.y=incidentRay.originalPoint.y+incidentRay.direction.y*t;
				 reflectedRay.originalPoint.z=incidentRay.originalPoint.z+incidentRay.direction.z*t;
				 Point n;
				 n.x=b2*c3-b3*c2;n.y=b3*c1-b1*c3;n.z=b1*c2-c1*b2;
				 RADC_UitizePoint(&n.x,&n.y,&n.z);
				 reflectedRay.direction=RADC_GetReflectedPoint(incidentRay.direction,n);
				 *reflectedFace=i;
				 if (u>0&&u<1&&(u+v)>0&&(u+v)<1)
				 {
					 *flag=1;
				 }
			 }
		 }
	 }
	 return reflectedRay;
 }



 __device__ bool JudgeTwoTriangleFacesOnTheSameQuadrangle(int faceNumber1,int faceNumber2)
 {
	 int a=faceNumber1%12;
	 int b=faceNumber2%12;
	 if (faceNumber1==faceNumber2)
	 {
		 return true;
	 }
	 if (a%2==0)
	 {
		 if (b==a+1)
		 {
			 return  true;
		 }
	 }
	 if (a%2==1)
	 {
		 if (b==a-1)
		 {
			 return true;
		 }
	 }
	 return false;
 }

 __device__ bool JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(int faceNumber1,int faceNumber2)
 {
	 int a=faceNumber1%12;
	 int b=faceNumber2%12;
	 if (a%2==0&&a!=0)
	 {
		 if (b==a-1)
		 {
			 return  true;
		 }
	 }
	 if (a%2==1&&a!=7)
	 {
		 if (b==a+1)
		 {
			 return true;
		 }
	 }
	 if ((a==0&&b==7)||(a==7&&b==0))
	 {
		 return true;
	 }
	 return false;
 }

 __device__ int GetRemainingOneNumber(int a,int b)
 {
	 return 3-(a+b);
 }

 __device__ void GetRemainingTwoNumber(int a,int *b,int *c)
 {
	 if (a==0)
	 {
		 *b=1;
		 *c=2;
	 }
	 if (a==1)
	 {
		 *b=0;
		 *c=2;
	 }
	 if (a==2)
	 {
		 *b=0;
		 *c=1;
	 }
 }

 __device__ void GetReflectedRayTubeWhenOneReflectedRayExist(int flagNumber,Ray *incidentRays,TriangleRayTube *reflectedRayTubes,Ray *reflectedRays,int *reflectedFace,Face *face,int i,int *reflectionExist,int *diffractionExist,int *faceNumber1,int *faceNumber2)
 {
	 int tempFace,tempFlag;
	 int remainingNumber1,int remainingNumber2;
	 GetRemainingTwoNumber(flagNumber,&remainingNumber1,&remainingNumber2);
	 reflectedRayTubes[2*i].ray1=reflectedRays[flagNumber];
	 reflectedRayTubes[2*i].ray2=CalculateReflectedRay(incidentRays[remainingNumber1],&face[reflectedFace[flagNumber]],1,&tempFace,&tempFlag);
	 reflectedRayTubes[2*i].ray3=CalculateReflectedRay(incidentRays[remainingNumber2],&face[reflectedFace[flagNumber]],1,&tempFace,&tempFlag);
	 reflectedRayTubes[2*i+1].ray1=reflectedRays[flagNumber];
	 reflectedRayTubes[2*i+1].ray2=CalculateReflectedRay(incidentRays[remainingNumber1],&face[reflectedFace[flagNumber]],1,&tempFace,&tempFlag);
	 reflectedRayTubes[2*i+1].ray3=CalculateReflectedRay(incidentRays[remainingNumber2],&face[reflectedFace[flagNumber]],1,&tempFace,&tempFlag);
	 diffractionExist[i]=1;
	 reflectionExist[i]=1;
	 faceNumber1[i]=reflectedFace[flagNumber];
	 faceNumber2[i]=reflectedFace[flagNumber];
 }

 __device__ void GetReflectedRayTubeWhenTwoReflectedRayExist(int flagNumber1,int flagNumber2,Ray *incidentRays,TriangleRayTube *reflectedRayTubes,Ray *reflectedRays,int *reflectedFace,Face *face,int i,int *reflectionExist,int *diffractionExist,int *faceNumber1,int *faceNumber2)
 {
	 int tempFace,tempflag=0;
	 int remainingNumber=GetRemainingOneNumber(flagNumber1,flagNumber2);
	 reflectedRayTubes[2*i].ray1=reflectedRays[flagNumber1];
	 reflectedRayTubes[2*i].ray2=CalculateReflectedRay(incidentRays[flagNumber2],&face[reflectedFace[flagNumber1]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i].ray3=CalculateReflectedRay(incidentRays[remainingNumber],&face[reflectedFace[flagNumber1]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i+1].ray1=reflectedRays[flagNumber2];
	 reflectedRayTubes[2*i+1].ray2=CalculateReflectedRay(incidentRays[flagNumber1],&face[reflectedFace[flagNumber2]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i+1].ray3=CalculateReflectedRay(incidentRays[remainingNumber],&face[reflectedFace[flagNumber2]],1,&tempFace,&tempflag);
	 diffractionExist[i]=1;
	 reflectionExist[i]=1;
	 faceNumber1[i]=reflectedFace[flagNumber1];
	 faceNumber2[i]=reflectedFace[flagNumber2];
 }

 __device__ void GetReflectedRayTubeWhenThreeReflectedRayExist(int flagNumber1,int flagNumber2,Ray *incidentRays,TriangleRayTube *reflectedRayTubes,Ray *reflectedRays,int *reflectedFace,Face *face,int i,int *reflectionExist,int *diffractionExist,int *faceNumber1,int *faceNumber2)
 {
	 int tempFace,tempflag=0;
	 int remainingNumber=GetRemainingOneNumber(flagNumber1,flagNumber2);
	 reflectedRayTubes[2*i].ray1=reflectedRays[flagNumber1];
	 reflectedRayTubes[2*i].ray2=reflectedRays[flagNumber2];
	 reflectedRayTubes[2*i].ray3=CalculateReflectedRay(incidentRays[remainingNumber],&face[reflectedFace[flagNumber1]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i+1].ray1=CalculateReflectedRay(incidentRays[flagNumber1],&face[reflectedFace[remainingNumber]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i+1].ray2=CalculateReflectedRay(incidentRays[flagNumber2],&face[reflectedFace[remainingNumber]],1,&tempFace,&tempflag);
	 reflectedRayTubes[2*i+1].ray3=reflectedRays[remainingNumber];
	 diffractionExist[i]=1;
	 reflectionExist[i]=1;
	 if (JudgeTwoTriangleFacesOnTheAdjacentQuadrangle(reflectedFace[remainingNumber],reflectedFace[flagNumber1]))
	 {
		 faceNumber1[i]=reflectedFace[flagNumber1];
	 }
	 else
	 {
		 faceNumber1[i]=reflectedFace[flagNumber2];
	 }
	 faceNumber2[i]=reflectedFace[remainingNumber];
 }

 __global__ void CalculateReflectionAndDiffractionWithBuildingFace(TriangleRayTube *incidentRayTubes,int faceCount,int rayTubeCount,TriangleRayTube *reflectedRayTubes,Face *face,Edge *diffractionEdge,int *faceNumber1,int *faceNumber2,int *reflectionExist,int *diffractionExist)
 {
	int i=blockIdx.x * blockDim.x+threadIdx.x;
	int reflectedFace[3];
	int flag[3]={0,0,0};
	Ray reflectedRays[3],incidentRays[3];
	incidentRays[0]=incidentRayTubes[i].ray1;
	incidentRays[0]=incidentRayTubes[i].ray2;
	incidentRays[0]=incidentRayTubes[i].ray3;
	reflectedRays[0]=CalculateReflectedRay(incidentRays[0],face,faceCount,&reflectedFace[0],&flag[0]);
	reflectedRays[1]=CalculateReflectedRay(incidentRays[1],face,faceCount,&reflectedFace[1],&flag[1]);
	reflectedRays[2]=CalculateReflectedRay(incidentRays[2],face,faceCount,&reflectedFace[2],&flag[2]);
	if (flag[0]+flag[1]+flag[2]==0)
	{
		reflectionExist[i]=0;
		diffractionExist[i]=0;
		faceNumber1[i]=0;
		faceNumber2[i]=0;
	}
	if (flag[0]+flag[1]+flag[2]==1)
	{
		for (int j=0;j<3;j++)
		{
			if (flag[j]==1)
			{
				GetReflectedRayTubeWhenOneReflectedRayExist(j,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
		}
	}
	if (flag[0]+flag[1]+flag[2]==2)
	{
		if (flag[0]==0)
		{
			if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[1],reflectedFace[2]))
			{
				GetReflectedRayTubeWhenOneReflectedRayExist(1,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
			else
			{
				GetReflectedRayTubeWhenTwoReflectedRayExist(1,2,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
		}
		if (flag[1]==0)
		{
			if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[2]))
			{
				GetReflectedRayTubeWhenOneReflectedRayExist(0,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
			else
			{
				GetReflectedRayTubeWhenTwoReflectedRayExist(0,2,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
			
		}
		if (flag[2]==0)
		{
			if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1]))
			{
				GetReflectedRayTubeWhenOneReflectedRayExist(0,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}
			else
			{
				GetReflectedRayTubeWhenTwoReflectedRayExist(0,1,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
			}			
	
		}
	}
	if (flag[0]+flag[1]+flag[2]==3)
	{
		if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1])&&JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[2]))
		{
			reflectedRayTubes[2*i].ray1=reflectedRays[0];
			reflectedRayTubes[2*i].ray2=reflectedRays[1];
			reflectedRayTubes[2*i].ray3=reflectedRays[2];
			reflectedRayTubes[2*i+1].ray1=reflectedRays[0];
			reflectedRayTubes[2*i+1].ray2=reflectedRays[1];
			reflectedRayTubes[2*i+1].ray3=reflectedRays[2];
			reflectionExist[i]=1;

			diffractionExist[i]=0;
		}
		if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1])&&(!JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[2])))
		{
			GetReflectedRayTubeWhenThreeReflectedRayExist(0,1,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
		}
		if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[2])&&(!JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1])))
		{
			GetReflectedRayTubeWhenThreeReflectedRayExist(0,2,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
		}
		if (JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[1],reflectedFace[2])&&(!JudgeTwoTriangleFacesOnTheSameQuadrangle(reflectedFace[0],reflectedFace[1])))
		{
			GetReflectedRayTubeWhenThreeReflectedRayExist(1,2,incidentRays,reflectedRayTubes,reflectedRays,reflectedFace,face,i,reflectionExist,diffractionExist,faceNumber1,faceNumber2);
		}
	}
}



hipError_t  GetReflectionAndDiffractionWithBuildingFace(TriangleRayTube *incidentRayTubes,int faceCount,int rayTubeCount,Face *buildingFace,TriangleRayTube *reflectedRayTubes,Edge *diffractionEdge,int *faceNumber1,int *faceNumber2,int *diffractionExist,int *reflectionExist)
{
	const int pointPerFace=9;
	hipError_t cudaStatus;

	TriangleRayTube *device_incidentRayTubes=0;
	TriangleRayTube *device_reflectedRayTubes=0;
	Edge *device_edge=0;
	Face *device_face=0;
	//float *device_distance=0;
	int *device_faceNumber1=0;
	int *device_faceNumber2=0;
	int *device_diffractionExist=0;
	int *device_reflectionExist=0;
	cudaStatus=hipSetDevice(0);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"CUDA capable GPU is not available!");
		goto Error;
	}

	cudaStatus=hipMalloc((void**)&device_incidentRayTubes,rayTubeCount*sizeof(TriangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_incidentRayTubes hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_reflectedRayTubes,rayTubeCount*2*sizeof(TriangleRayTube));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_reflectedRayTubes hipMalloc error!");

		goto Error;
	}

	cudaStatus=hipMalloc((void**)&device_edge,rayTubeCount*sizeof(Edge));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_edge hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_face,faceCount*sizeof(Face));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_face hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_faceNumber1,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_faceNumber1 hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_faceNumber2,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_faceNumber2 hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_diffractionExist,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_diffractionExist hipMalloc error!");
		goto Error;
	}
	cudaStatus=hipMalloc((void**)&device_reflectionExist,rayTubeCount*sizeof(int));
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"device_reflectionExis hipMalloc error!");
		goto Error;
	}

	cudaStatus=hipMemcpy(device_incidentRayTubes,incidentRayTubes,rayTubeCount*sizeof(TriangleRayTube),hipMemcpyHostToDevice);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"incidentRayTubes Memcpy failed!");
	}
	cudaStatus=hipMemcpy(device_face,buildingFace,faceCount*sizeof(Face),hipMemcpyHostToDevice);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"face Memcpy failed!");
	}

	const int num_blocks=32;
	const int num_threads=640;

	CalculateReflectionAndDiffractionWithBuildingFace<<<num_blocks,num_threads>>>(device_incidentRayTubes,faceCount,rayTubeCount,device_reflectedRayTubes,device_face,device_edge,device_faceNumber1,device_faceNumber2,device_reflectionExist,device_diffractionExist);

	cudaStatus=hipMemcpy(reflectedRayTubes,device_reflectedRayTubes,rayTubeCount*2*sizeof(TriangleRayTube),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"reflectedRayTubes hipMemcpy failed!");
	}
	cudaStatus=hipMemcpy(faceNumber1,device_faceNumber1,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"faceNumber1 hipMemcpy failed!");
	}
	cudaStatus=hipMemcpy(faceNumber2,device_faceNumber2,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"faceNumber2 hipMemcpy failed!");
	}
	cudaStatus=hipMemcpy(diffractionEdge,device_edge,rayTubeCount*sizeof(Edge),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionEdge hipMemcpy failed!");
	}
	cudaStatus=hipMemcpy(reflectionExist,device_reflectionExist,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"reflectionExist hipMemcpy failed!");
	}
	cudaStatus=hipMemcpy(diffractionExist,device_diffractionExist,rayTubeCount*sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"diffractionExist hipMemcpy failed!");
	}
	return cudaStatus;
Error:
	hipFree(device_incidentRayTubes);
	hipFree(device_reflectedRayTubes);
	hipFree(device_edge);
	hipFree(device_face);
	hipFree(device_faceNumber1);
	hipFree(device_faceNumber2);
	hipFree(device_reflectionExist);
	hipFree(device_diffractionExist);
}